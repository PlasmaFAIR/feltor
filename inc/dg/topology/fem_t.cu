#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cusp/print.h>
#include "dg/algorithm.h"
#include "fem.h"
#include "fem_weights.h"

double function( double x, double y){return sin(x)*cos(y);}

using Vector = dg::DVec;
using Matrix = cusp::coo_matrix<int,double,cusp::device_memory>;
using MassMatrix = dg::KroneckerTriDiagonal2d<Vector>;
using InvMassMatrix = dg::InverseKroneckerTriDiagonal2d<Vector>;

int main ()
{
    unsigned n = 3, Nx = 18, Ny = 24, mx = 3;
    double eps = 1e-10;
    //std::cout << "# Type in n Nx Ny mx eps!\n";
    std::cout << "# on grid " << n <<" x "<<Nx<<" x "<<Ny<<"\n";
    std::cout << "# eps and Multiply " << eps <<" " << mx<<"\n";
    dg::CartesianGrid2d gDIR( 0, 2.*M_PI, M_PI/2., 5*M_PI/2., n, Nx, Ny, dg::DIR,
            dg::DIR);
    dg::FemRefinement fem_ref( mx);
    dg::CartesianRefinedGrid2d gDIR_f( fem_ref, fem_ref, gDIR.x0(), gDIR.x1(),
            gDIR.y0(), gDIR.y1(), n, Nx,Ny, dg::DIR, dg::DIR);
    const Vector func = dg::evaluate( function, gDIR);
    const Vector v2d = dg::create::fem_inv_weights( gDIR);
    const Vector w2d = dg::create::fem_weights( gDIR);
    const Vector wf2d = dg::create::volume( gDIR_f);
    double integral = dg::blas2::dot( func, w2d, func);
    std::cout << "error of integral is "
              <<(integral-M_PI*M_PI)/M_PI/M_PI<<std::endl;
    dg::HVec Xf = dg::pullback( dg::cooX2d, gDIR_f);
    dg::HVec Yf = dg::pullback( dg::cooY2d, gDIR_f);
    Matrix inter = dg::create::interpolation( Xf, Yf, gDIR, dg::NEU, dg::NEU, "linear");
    Matrix interT = dg::transpose( inter);
    Matrix Wf = dg::create::diagonal( (dg::HVec)wf2d), project;
    Matrix Vf = dg::create::diagonal( (dg::HVec)v2d), tmp;
    cusp::multiply( interT, Wf, tmp);
    cusp::multiply( Vf, tmp, project);
    project.sort_by_row_and_column();
    //std::cout << "Project matrix\n";
    //cusp::print( project);


    Vector func_f( gDIR_f.size());
    dg::blas2::symv( inter, func, func_f);
    integral = dg::blas2::dot( func_f, wf2d, func_f);
    std::cout << "error of refined integral is "
              <<(integral-M_PI*M_PI)/M_PI/M_PI<<std::endl;
    Vector barfunc(func);
    dg::blas2::symv( project, func_f, barfunc);
    // test now should contain Sf
    Vector test( barfunc);
    dg::PCG<Vector> cg( test, 1000);
    // PCG tests fem-mass
    MassMatrix fem_mass = dg::create::fem_mass( gDIR);
    //std::cout << "S matrix\n";
    //cusp::print( fem_mass);
    unsigned number = cg.solve( fem_mass, test, barfunc, 1., w2d, eps);
    dg::blas1::axpby( 1., func, -1., test);
    double norm = sqrt(dg::blas2::dot( w2d, test) );
    double func_norm = sqrt(dg::blas2::dot( w2d, func) );
    std::cout <<"PCG Distance to true solution: "<<norm/func_norm<<"\n";
    std::cout << "using "<<number<<" iterations\n";
    InvMassMatrix inv_fem_mass = dg::create::inv_fem_mass( gDIR);
    dg::blas2::symv( inv_fem_mass, barfunc, test);
    dg::blas1::axpby( 1., func, -1., test);
    norm = sqrt(dg::blas2::dot( w2d, test) );
    std::cout <<"Thomas Distance to true solution: "<<norm/func_norm<<"\n";


    std::cout << "TEST L2C projection\n";
    Matrix interC = dg::create::interpolation( Xf, Yf, gDIR, dg::NEU, dg::NEU, "nearest");
    interT = dg::transpose( interC);
    cusp::multiply( interT, Wf, tmp);
    cusp::multiply( Vf, tmp, project);
    project.sort_by_row_and_column();
    dg::blas2::symv( project, func_f, barfunc);
    fem_mass = dg::create::fem_linear2const( gDIR);

    number = cg.solve( fem_mass, test, barfunc, 1., w2d, eps);
    dg::blas1::axpby( 1., func, -1., test);
    norm = sqrt(dg::blas2::dot( w2d, test) );
    std::cout <<"PCG Distance to true solution: "<<norm/func_norm<<"\n";
    std::cout << "using "<<number<<" iterations\n";
    inv_fem_mass = dg::create::inv_fem_linear2const( gDIR);
    dg::blas2::symv( inv_fem_mass, barfunc, test);
    dg::blas1::axpby( 1., func, -1., test);
    norm = sqrt(dg::blas2::dot( w2d, test) );
    std::cout <<"Thomas Distance to true solution: "<<norm/func_norm<<"\n";

    return 0;
}
