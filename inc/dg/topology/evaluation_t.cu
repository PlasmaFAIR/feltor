#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "dg/blas.h"
#include "dg/functors.h"

#include "evaluation.h"
#include "weights.h"

struct exp_function{
DG_DEVICE
double operator()( double x)
{
    return exp(x);
}
};
struct sin_function{
DG_DEVICE
double operator()( double x)
{
    return sin(x);
}
};
template<class T>
T function(T x, T y)
{
    T rho = 0.20943951023931953; //pi/15
    T delta = 0.050000000000000003;
    if( y<= M_PI)
        return delta*cos(x) - 1./rho/cosh( (y-M_PI/2.)/rho)/cosh( (y-M_PI/2.)/rho);
    return delta*cos(x) + 1./rho/cosh( (3.*M_PI/2.-y)/rho)/cosh( (3.*M_PI/2.-y)/rho);
}
double function3d( double x, double y, double z)
{
        return exp(x)*exp(y)*exp(z);
}


int main()
{
    std::cout << "This program tests the exblas::dot function. The tests succeed only if the evaluation and grid functions but also the weights and especially the exblas::dot function are correctly implemented and compiled. Furthermore, the compiler implementation of the exp function in the math library must be consistent across platforms to get reproducible results\n";
    std::cout << "A TEST is PASSED if the number in the second column shows EXACTLY 0!\n";
    unsigned n = 3, Nx = 12, Ny = 28, Nz = 100;
    std::cout << "On Grid "<<n<<" x "<<Nx<<" x "<<Ny<<" x "<<Nz<<"\n";

    dg::Grid1d g1d( 1, 2, n, 12);
    dg::Grid2d g2d( 0.0, 6.2831853071795862, 0.0, 6.2831853071795862, 3, 48, 48);
    dg::RealGrid2d<float> gf2d( 0.0, 6.2831853071795862, 0.0, 6.2831853071795862, 3, 48, 48);
    dg::Grid3d g3d( 1, 2, 3, 4, 5, 6, n, Nx, Ny, Nz,dg::PER,dg::PER,dg::PER);

    //test evaluation functions
    const dg::DVec func1d = dg::construct<dg::DVec>( dg::evaluate( exp, g1d));
    const dg::DVec func2d = dg::construct<dg::DVec>( dg::evaluate( function<double>, g2d));
    const dg::fDVec funcf2d = dg::construct<dg::fDVec>( dg::evaluate( function<float>, gf2d));
    const dg::DVec func3d = dg::construct<dg::DVec>( dg::evaluate( function3d, g3d));
    const dg::DVec w1d = dg::construct<dg::DVec>( dg::create::weights( g1d));
    const dg::DVec w2d = dg::construct<dg::DVec>( dg::create::weights( g2d));
    const dg::fDVec wf2d = dg::construct<dg::fDVec>( dg::create::weights( gf2d));
    const dg::DVec w3d = dg::construct<dg::DVec>( dg::create::weights( g3d));
    dg::exblas::udouble res;

    double integral = dg::blas1::dot( w1d, func1d); res.d = integral;
    std::cout << "1D integral               "<<std::setw(6)<<integral <<"\t" << res.i - 4616944842743393935  << "\n";
    double sol = (exp(2.) -exp(1));
    std::cout << "Correct integral is       "<<std::setw(6)<<sol<<std::endl;
    std::cout << "Relative 1d error is      "<<(integral-sol)/sol<<"\n\n";

    double integral2d = dg::blas1::dot( w2d, func2d); res.d = integral2d;
    std::cout << "2D integral               "<<std::setw(6)<<integral2d <<"\t" << res.i + 4823280491526356992<< "\n";
    double sol2d = 0;
    std::cout << "Correct integral is       "<<std::setw(6)<<sol2d<<std::endl;
    std::cout << "2d error is               "<<(integral2d-sol2d)<<"\n\n";

    float integralf2d = dg::blas1::dot( wf2d, funcf2d); res.d = integralf2d;
    std::cout << "2D integral (float)       "<<std::setw(6)<<integralf2d <<"\n";
    float solf2d = 0;
    std::cout << "Correct integral is       "<<std::setw(6)<<solf2d<<std::endl;
    std::cout << "2d error (float)          "<<(integralf2d-solf2d)<<"\n\n";

    double integral3d = dg::blas1::dot( w3d, func3d); res.d = integral3d;
    std::cout << "3D integral               "<<std::setw(6)<<integral3d <<"\t" << res.i - 4675882723962622631<< "\n";
    double sol3d = (exp(2.)-exp(1))*(exp(4.)-exp(3))*(exp(6.)-exp(5));
    std::cout << "Correct integral is       "<<std::setw(6)<<sol3d<<std::endl;
    std::cout << "Relative 3d error is      "<<(integral3d-sol3d)/sol3d<<"\n\n";

    double norm = dg::blas2::dot( func1d, w1d, func1d); res.d = norm;
    std::cout << "Square normalized 1D norm "<<std::setw(6)<<norm<<"\t" << res.i - 4627337306989890294 <<"\n";
    double solution = (exp(4.) -exp(2))/2.;
    std::cout << "Correct square norm is    "<<std::setw(6)<<solution<<std::endl;
    std::cout << "Relative 1d error is      "<<(norm-solution)/solution<<"\n\n";

    double norm2d = dg::blas2::dot( w2d, func2d); res.d = norm2d;
    std::cout << "Square normalized 2D norm "<<std::setw(6)<<norm2d<<"\t" << res.i - 4635333359953759707<<"\n";
    double solution2d = 80.0489;
    std::cout << "Correct square norm is    "<<std::setw(6)<<solution2d<<std::endl;
    std::cout << "Relative 2d error is      "<<(norm2d-solution2d)/solution2d<<"\n\n";

    double norm3d = dg::blas2::dot( func3d, w3d, func3d); res.d = norm3d;
    std::cout << "Square normalized 3D norm "<<std::setw(6)<<norm3d<<"\t" << res.i - 4746764681002108278<<"\n";
    double solution3d = (exp(4.)-exp(2))/2.*(exp(8.)-exp(6.))/2.*(exp(12.)-exp(10))/2.;
    std::cout << "Correct square norm is    "<<std::setw(6)<<solution3d<<std::endl;
    std::cout << "Relative 3d error is      "<<(norm3d-solution3d)/solution3d<<"\n\n";

    std::cout << "TEST result of a sin and exp function to compare compiler specific math libraries:\n";
    dg::DVec x(1, 6.12610567450009658);
    dg::blas1::transform( x, x, sin_function() );
    res.d = x[0];
    std::cout << "Result of sin:    "<<res.i<<"\n"
              << "          GCC:    -4628567870976535683 (correct)"<<std::endl;
    dg::DVec y(1, 5.9126151457310376);
    dg::blas1::transform( y, y, exp_function() );
    res.d = y[0];
    std::cout << "Result of exp:     "<<res.i<<"\n"
              << "          GCC:     4645210948416067678 (correct)"<<std::endl;

    //TEST OF INTEGRAL
    dg::HVec integral_num = dg::integrate( cos, g1d);
    dg::HVec integral_ana = dg::evaluate( sin, g1d);
    dg::blas1::plus( integral_ana, -sin(g1d.x0()));
    dg::blas1::axpby( 1., integral_ana, -1., integral_num);
    norm = dg::blas2::dot( integral_num, dg::create::weights( g1d), integral_num);
    std::cout << " Error norm of  1d integral function "<<norm<<"\n";
    // TEST if dot throws on NaN
    std::cout << "TEST if dot throws on Inf or Nan:\n";
    dg::blas1::transform( x,x, dg::LN<double>());
    thrust::device_vector<bool> boolvec ( 100, false);
    dg::blas1::transform( x, boolvec, dg::ISNFINITE<double>());
    bool hasnan = dg::blas1::reduce( boolvec, false, thrust::logical_or<bool>());
    std::cout << "x contains Inf or Nan numbers "<<std::boolalpha<<hasnan<<"\n";
    try{
        dg::blas1::dot( x,x);
    }catch ( std::exception& e)
    {
        std::cerr << "Error thrown as expected\n";
        //std::cerr << e.what() << std::endl;
    }
    std::cout << "Test MinMod function:\n";
    dg::MinMod minmod;
    std::cout << " 3 -5 ="<<minmod( 3,-5)<< " (0) "<<std::endl;
    std::cout << " 2 4 1 ="<<minmod( 2,4,1)<< " (1) "<<std::endl;
    std::cout << " 0 1 2 ="<<minmod( 0,1,2)<< " (0) "<<std::endl;
    std::cout << " -1 1 2 ="<<minmod( -1,1,2)<< " (0) "<<std::endl;
    std::cout << " -5 -3 -2 ="<<minmod( -5,-3,-2)<< " (-2) "<<std::endl;

    std::cout << "\nFINISHED! Continue with topology/derivatives_t.cu !\n\n";
    return 0;
}
