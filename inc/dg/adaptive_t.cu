#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <functional>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "backend/typedefs.h"
#include "topology/evaluation.h"
#include "topology/grid.h"
#include "arakawa.h"
#include "runge_kutta.h"
#include "adaptive.h"


//![function]
struct RHS
{
    RHS( double damping, double omega_0, double omega_drive) :
        m_d( damping), m_w0( omega_0), m_wd( omega_drive){}
    void operator()( double t, const std::array<double,2>& y,
            std::array<double,2>& yp)
    {
        //damped driven harmonic oscillator
        // x -> y[0] , v -> y[1]
        yp[0] = y[1];
        yp[1] = -2.*m_d*m_w0*y[1] - m_w0*m_w0*y[0] + sin(m_wd*t);
    }
    void operator()( double alpha, double t, std::array<double,2>& y,
            const std::array<double,2>& yp)
    {
        // y - alpha RHS( t, y) = rho
        // can be solved analytically
        y[1] = ( yp[1] + alpha*sin(m_wd*t) - alpha* m_w0*m_w0*yp[0])/
               (1.+2.*alpha*m_d*m_w0+alpha*alpha*m_w0*m_w0);
        y[0] = yp[0] + alpha*y[1];
    }
    private:
    double m_d, m_w0, m_wd;
};
//![function]

std::array<double, 2> solution( double t, double damping, double omega_0,
        double omega_drive)
{
    double tmp1 = (2.*omega_0*damping);
    double tmp2 = (omega_0*omega_0 - omega_drive*omega_drive)/omega_drive;
    double amp = 1./sqrt( tmp1*tmp1 + tmp2*tmp2);
    double phi = atan( 2.*omega_drive*omega_0*damping/
            (omega_drive*omega_drive-omega_0*omega_0));

    double x = amp*sin(omega_drive*t+phi)/omega_drive;
    double v = amp*cos(omega_drive*t+phi);
    return {x,v};
}

int main()
{
    std::cout << "Program to test correct implementation of adaptive methods in adaptive.h at the example of the damped driven harmonic oscillator. Errors should be small! \n";
    std::cout << std::scientific;
    //![doxygen]
    //... in main
    //set start and end time
    double t_start = 0., t_end = 1.;
    //set physical parameters and initial condition
    const double damping = 0.2, omega_0 = 1.0, omega_drive = 0.9;
    std::array<double,2> u_start = solution(t_start, damping, omega_0,
            omega_drive), u_end(u_start);
    //construct a rhs with the right interface
    RHS rhs( damping, omega_0, omega_drive);
    double dt= 0;
    //integration
    int counter = dg::integrateERK( "Dormand-Prince-7-4-5", rhs, t_start,
            u_start, t_end, u_end, dt, dg::pid_control, dg::l2norm, 1e-6);
    //now compute error
    dg::blas1::axpby( 1., solution(t_end, damping, omega_0, omega_drive), -1.,
            u_end);
    std::cout << "With "<<counter<<"\t Dormand Prince steps norm of error is "
              << dg::l2norm( u_end)<<"\n";
    //![doxygen]
    std::cout << "Explicit Methods \n";
    std::vector<std::string> names{
        "Heun-Euler-2-1-2",
        "Cavaglieri-3-1-2 (explicit)",
        "Fehlberg-3-2-3",
        "Fehlberg-4-2-3",
        "Bogacki-Shampine-4-2-3",
        "Cavaglieri-4-2-3 (explicit)",
        "ARK-4-2-3 (explicit)",
        "Zonneveld-5-3-4",
        "ARK-6-3-4 (explicit)",
        "Sayfy-Aburub-6-3-4",
        "Cash-Karp-6-4-5",
        "Fehlberg-6-4-5",
        "Dormand-Prince-7-4-5",
        "Tsitouras09-7-4-5",
        "Tsitouras11-7-4-5",
        "ARK-8-4-5 (explicit)",
        "Verner-9-5-6",
        "Verner-10-6-7",
        "Fehlberg-13-7-8",
        "Dormand-Prince-13-7-8",
        "Feagin-17-8-10"
    };
    for( auto name : names)
    {
        dt = 0;
        u_start = solution(t_start, damping, omega_0, omega_drive);
        counter = dg::integrateERK( name, rhs, t_start, u_start, t_end,
                u_end, dt, dg::pid_control, dg::l2norm, 1e-6, 1e-10);

        std::array<double, 2> sol = solution(t_end, damping, omega_0, omega_drive);
        dg::blas1::axpby( 1.,sol  , -1., u_end);
        std::cout << "With "<<std::setw(6)<<counter<<" steps norm of error in "
                  <<std::setw(24)<<name<<"\t"<<dg::l2norm( u_end)<<"\n";
    }
    ///-------------------------------Implicit Methods----------------------//
    std::cout << "Implicit Methods \n";
    std::vector<std::string> implicit_names{
        "SDIRK-2-1-2",
        "Cavaglieri-3-1-2 (implicit)",
        "Billington-3-3-2",
        "TRBDF2-3-3-2",
        "Kvaerno-4-2-3",
        "Cavaglieri-4-2-3 (implicit)",
        "ARK-4-2-3 (implicit)",
        "Cash-5-2-4",
        "Cash-5-3-4",
        "SDIRK-5-3-4",
        "ARK-6-3-4 (implicit)",
        "Kvaerno-7-4-5",
        "ARK-8-4-5 (implicit)",
    };
    for( auto name : implicit_names)
    {
        dt = 0;
        u_start = solution(t_start, damping, omega_0, omega_drive);
        dg::Adaptive< dg::DIRKStep< std::array<double,2> > >
                pd( name, u_start);
        auto stepper = [&](double& t, std::array<double,2>& u, double& dt)
        {
            pd.step( rhs, rhs, t, u, t, u, dt, dg::im_control, dg::l2norm, 1e-6, 1e-10);
        };
        counter = dg::integrate( stepper, t_start, u_start, t_end,
            u_end, dt);

        std::array<double, 2> sol = solution(t_end, damping, omega_0, omega_drive);
        dg::blas1::axpby( 1.,sol  , -1., u_end);
        std::cout << "With "<<std::setw(6)<<counter<<" steps norm of error in "
                  <<std::setw(24)<<name<<"\t"<<dg::l2norm( u_end)<<"\n";
    }
    ///---------------------------Test domain restriction-------------------//
    std::cout << "Test domain restriction \n";
    for( auto name : names)
    {
        double dt = 0;
        double t_start = 0;
        double t_end = 10;
        double u_start = 1.0, u_end;
        auto rhs = [](double t, double y, double& yp){
                yp = y;
        };
        dg::Adaptive<dg::ERKStep<double>> pd( name,u_start);
        auto adapt = [&](double& t, double& u, double& dt)
        {
            pd.step( rhs, t, u, t, u, dt, dg::pid_control, dg::l2norm, 1e-6, 1e-10);
        };
        unsigned counter = dg::integrate_in_domain( adapt , t_start, u_start, t_end,
                u_end, dt, dg::Grid1d( 0., 100., 1,1), 1e-4  );
        double analytic = log( 100.);
        std::cout << "With "<<std::setw(6)<<counter<<" steps norm of error in "
                  <<std::setw(24)<<name<<"\t"<<fabs( t_end - analytic)<<"\n";
    }
    return 0;
}
