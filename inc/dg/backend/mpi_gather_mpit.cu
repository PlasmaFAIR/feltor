#include "hip/hip_runtime.h"
#include <iostream>

#include <random> // for random shuffle
#include <mpi.h>
#include "../blas1.h"
#include "mpi_gather.h"

template<class T>
bool is_equal( const T& v, const T& w)
{
    bool equal = true;
    int rank;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    for( unsigned i=0; i<v.size(); i++)
    {
        if( v[i] != w[i])
        {
            std::cout << rank<<" "<<v[i] << " "<<w[i]<<"\n";
            equal = false;
        }
    }
    return equal;
}

template<template<class> class Vector, class value_type>
void gather_test( const thrust::host_vector<std::array<int,2>>& gIdx,
    const Vector<value_type>& v,
    const Vector<value_type>& ana, bool bijective = false
    )
{
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
    thrust::host_vector<int> bufferIdx;
    auto recv_map = dg::gIdx2unique_idx( gIdx, bufferIdx);
    dg::MPIGather<Vector> mpi_gather(recv_map, MPI_COMM_WORLD);
    dg::LocalGatherMatrix<Vector> local_gather(bufferIdx);
    Vector<value_type> buffer( mpi_gather.buffer_size());
    mpi_gather.global_gather_init( v, buffer);
    mpi_gather.global_gather_wait( buffer);
    Vector<value_type> num(ana);
    local_gather.gather( buffer, num);
    bool equal  = is_equal( ana, num);
    std::cout <<"GATHER Rank "<<rank<< (equal ? " PASSED" : " FAILED")<<std::endl;
    if( bijective) // Scatter the index
    {
        auto sIdx = dg::mpi_invert_permutation( gIdx, MPI_COMM_WORLD);
        auto recv_map = dg::gIdx2unique_idx( sIdx, bufferIdx);
        dg::LocalGatherMatrix<Vector> local_gather(bufferIdx);

        dg::MPIGather<Vector > mpi_gather(recv_map, MPI_COMM_WORLD);
        num = v;
        dg::blas1::copy( 0, num);
        Vector<value_type> buffer( mpi_gather.buffer_size());
        mpi_gather.global_gather_init( ana, buffer);
        mpi_gather.global_gather_wait( buffer);
        local_gather.gather( buffer, num);
        equal  = is_equal( v, num);
        if(!equal)std::cout <<"SCATTER Rank "<<rank<<" FAILED"<<std::endl;
    }
}
template<class value_type>
void mpi_gather_test( const thrust::host_vector<std::array<int,2>>& gIdx,
    const thrust::host_vector<value_type>& v,
    const thrust::host_vector<value_type>& ana, bool bijective = false
    )
{
    int rank;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    // test mpi_gather
    auto num= ana;
    dg::mpi_gather( gIdx, v, num, MPI_COMM_WORLD);
    bool equal  = is_equal( ana, num);
    if(!equal) std::cout <<"MPI GATHER Rank "<<rank<< " FAILED"<<std::endl;
    if( bijective)
    {
        num = v;
        dg::blas1::copy( 0, num);
        dg::mpi_scatter( gIdx, ana, num, MPI_COMM_WORLD);
        equal  = is_equal( v, num);
        if(!equal) std::cout <<"MPI SCATTER Rank "<<rank<<" FAILED"<<std::endl;
    }
}

// If you get hipIpcCloseMemHandle failed errors when executing with cuda
// The cause the IN_PLACE option in mpi functions
// then https://github.com/horovod/horovod/issues/82
// --mca btl_smcuda_use_cuda_ipc 0
int main( int argc, char * argv[])
{
    MPI_Init( &argc, &argv);
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);

    if(rank==0)std::cout <<"# processes =  " <<size <<std::endl;
    // TODO Add random shuffle test

    {
    if(rank==0)std::cout << "Test: Simple shift (non-symmetric, bijective)\n";
    unsigned N=8, shift = 3, global_N = N*size;
    thrust::host_vector<double> v(N), ana(N);
    thrust::host_vector<std::array<int,2>> gIdx( N);
    for( unsigned i=0; i<gIdx.size(); i++)
    {
        gIdx[i][0] = i+shift >= N ? (rank+1)%size : rank; // PID
        gIdx[i][1] = (i + shift)%N; // local idx on PID
        v[i] = double( rank*N + i);
        ana[i] = double(( rank*N + i + shift) %global_N);
    }
    gather_test<thrust::device_vector, double>( gIdx, v, ana, true);
    mpi_gather_test( gIdx, v, ana, true);
    MPI_Barrier(MPI_COMM_WORLD);
    }
    {
    if(rank==0)std::cout << "Test simple transpose ( symmetric, bijective)"<<std::endl;

    unsigned global_rows = 2*size, global_cols = size;
    unsigned local_rows = global_rows, local_cols = 1;// matrix size  every rank holds a column
    unsigned local_rowsT = global_cols, local_colsT = 2;// matrix size  every rank holds a column
    // in the transposed matrix every rank holds 2 columns
    thrust::host_vector<double> mat( local_rows*local_cols ),
                                matT( local_rowsT*local_colsT);
    thrust::host_vector<std::array<int,2>> gIdx( local_rowsT*local_colsT);
    for( unsigned i=0; i<local_rows; i++)
    for( unsigned k=0; k<local_cols; k++)
        mat[i*local_cols + k] = (double)(i*global_cols + rank + k);
    for( unsigned i=0; i<local_rowsT; i++)
    for( unsigned k=0; k<local_colsT; k++)
    {
        gIdx[i*local_colsT+k][0] = i; // PID
        gIdx[i*local_colsT+k][1] = (rank*2 + k); // local idx
        matT[i*local_colsT+k] = double(rank*local_rowsT*local_colsT + k*local_rowsT+i);
    }
    gather_test<thrust::device_vector,double>( gIdx, mat, matT, true);
    mpi_gather_test( gIdx, mat, matT, true);
    MPI_Barrier(MPI_COMM_WORLD);
    }
    {
    if(rank==0)std::cout << " Repartition test with quadratic sizes ( bijective, inhomogeneous)\n";
    unsigned N=rank;
    unsigned NT = size-1-rank;
    thrust::host_vector<double> v(N), ana(NT);
    thrust::host_vector<std::array<int,2>> gIdx( NT);
    double value = double(rank*rank-rank)/2;
    for( unsigned i=0; i<N; i++)
        v[i] = value+i;
    int start = 0;
    for ( unsigned r=0; r<(unsigned)rank; r++)
    {
        int back_rank = size-1-r;
        start += back_rank;
    }
    for( unsigned i=0; i<NT; i++)
    {
        int globalIdx = start + i;
        int pid = floor( (1.+sqrt( 1.+globalIdx*8.))/2.);
        gIdx[i][0] = pid;
        gIdx[i][1] = globalIdx - (pid*pid-pid)/2;
        ana[i] = double( globalIdx);
    }
    gather_test<thrust::device_vector, double>( gIdx, v, ana, true);
    mpi_gather_test( gIdx, v, ana, true);
    MPI_Barrier(MPI_COMM_WORLD);

    }
    {
    if(rank==0)std::cout << " Random shuffle ( non-bijective, homogeneous)\n";
    unsigned N = 1000, global_N = size*N;
    thrust::host_vector<int> v(N), ana( N);
    thrust::host_vector<std::array<int,2>> gIdx( N);
    thrust::sequence( v.begin(), v.end());
    // The idea for a test is that if we gather the index we get the gather map

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> distr( 0, global_N-1);
    for( int i=0; i<(int)N; i++)
    {
        v[i] += rank*N; // we want the global index
        int idx = distr( gen);
        int pid = idx / N;
        int lIdx = idx % N;
        gIdx[i] = {pid, lIdx};
        ana[i] = idx;
    }

    std::cout << "RANK Random"<< rank<<"\n";
    for( unsigned u=0; u<10; u++)
        std::cout << gIdx[u][0]<<" "<<gIdx[u][1]<<" ";
    std::cout << std::endl;
    gather_test<thrust::device_vector, int>( gIdx, v, ana, false);
    mpi_gather_test( gIdx, v, ana, false);
    MPI_Barrier(MPI_COMM_WORLD);


    }


    MPI_Finalize();

    return 0;
}
