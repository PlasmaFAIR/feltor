#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>
#include "json/json.h"

#include "dg/algorithm.h"
#include "dg/file/nc_utilities.h"

#include "solovev.h"
#include "taylor.h"
//#include "guenter.h"
#include "curvilinearX.h"
#include "ribeiroX.h"
#include "ds.h"

double sine( double x) {return sin(x);}
double cosine( double x) {return cos(x);}

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::GridX3d& g)
{
    assert( g.Nz() == 2);
    thrust::host_vector<double> out(g.size());
    for( unsigned s=0; s<g.Nz(); s++)
    for( unsigned i=0; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((s*g.Ny()+i)*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];

    //exchange two segments
    for( unsigned i=g.outer_Ny(); i<2*g.outer_Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[(((i+g.inner_Ny())*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.inner_Ny()+g.outer_Ny(); i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[(((i-g.inner_Ny())*g.n() + k)*g.Nx() + j)*g.n()+l];
    if( g.outer_Ny() == 0)
    {
    //exchange two segments
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[(((i+1)*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[(((1*g.Ny() + i)*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[(((0)*g.n() + k)*g.Nx() + j)*g.n()+l];
    }


    return out;
}

int main( int argc, char* argv[])
{
    std::cout << "Type n, Nx, Ny, Nz (Nx must be divided by 4 and Ny by 10) \n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    Json::Value js;
    if( argc==1)
    {
        //std::ifstream is("geometry_params_Xpoint_taylor.json");
        std::ifstream is("geometry_params_Xpoint.json");
        is >> js;
    }
    else
    {
        std::ifstream is(argv[1]);
        is >> js;
    }
    dg::geo::solovev::Parameters gp(js);
    dg::Timer t;
    std::cout << "Type psi_0 \n";
    double psi_0 = -16;
    std::cin >> psi_0;
    std::cout << "Type fx and fy ( fx*Nx and fy*Ny must be integer) \n";
    double fx_0=1./4., fy_0=1./22.;
    std::cin >> fx_0>> fy_0;
    gp.display( std::cout);
    std::cout << "Constructing orthogonal grid ... \n";
    t.tic();
    dg::geo::CylindricalFunctorsLvl2 psip = dg::geo::solovev::createPsip(gp);
    std::cout << "Psi min "<<psip.f()(gp.R_0, 0)<<"\n";
    double R_X = gp.R_0-1.1*gp.triangularity*gp.a;
    double Z_X = -1.1*gp.elongation*gp.a;
    dg::geo::findXpoint( psip, R_X, Z_X);

    double R0 = gp.R_0, Z0 = 0;
    dg::geo::RibeiroX generator(psip, psi_0, fx_0, R_X,Z_X, R0, Z0);
    dg::geo::CurvilinearProductGridX3d g3d(generator, fx_0, fy_0, n, Nx, Ny,Nz, dg::DIR, dg::NEU);
    dg::geo::CurvilinearGridX2d g2d(generator, fx_0, fy_0, n, Nx, Ny, dg::DIR, dg::NEU);
    t.toc();
    dg::GridX3d g3d_periodic(g3d.x0(), g3d.x1(), g3d.y0(), g3d.y1(), g3d.z0(), g3d.z1(), g3d.fx(), g3d.fy(), g3d.n(), g3d.Nx(), g3d.Ny(), 2);
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    dg::Grid1d g1d( g2d.x0(), g2d.x1(), g2d.n(), g2d.Nx());
    dg::HVec x_left = dg::evaluate( sine, g1d), x_right(x_left);
    dg::HVec y_left = dg::evaluate( cosine, g1d);
    int ncid;
    dg::file::NC_Error_Handle err;
    err = nc_create( "ribeiroX.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[3], dim1d[1];
    err = dg::file::define_dimensions(  ncid, dim3d, g3d_periodic.grid());
    //err = dg::file::define_dimensions(  ncid, dim3d, g2d.grid());
    err = dg::file::define_dimension(  ncid, dim1d, g1d, "i");
    int coordsID[2], onesID, defID, volID, divBID;
    int coord1D[5];
    err = nc_def_var( ncid, "xc", NC_DOUBLE, 3, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "yc", NC_DOUBLE, 3, dim3d, &coordsID[1]);
    err = nc_def_var( ncid, "x_left", NC_DOUBLE, 1, dim1d, &coord1D[0]);
    err = nc_def_var( ncid, "y_left", NC_DOUBLE, 1, dim1d, &coord1D[1]);
    err = nc_def_var( ncid, "x_right", NC_DOUBLE, 1, dim1d, &coord1D[2]);
    err = nc_def_var( ncid, "y_right", NC_DOUBLE, 1, dim1d, &coord1D[3]);
    err = nc_def_var( ncid, "f_x", NC_DOUBLE, 1, dim1d, &coord1D[4]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 3, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 3, dim3d, &defID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 3, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 3, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip.f(), g2d);
    g2d.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g3d_periodic).data());
    //err = nc_put_var_double( ncid, onesID, periodify(g2d.g(), g3d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.map()[0][i];
        Y[i] = g2d.map()[1][i];
    }

    dg::DVec ones = dg::evaluate( dg::one, g2d);
    dg::DVec temp0( g2d.size()), temp1(temp0);
    dg::DVec w2d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g3d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g3d_periodic).data());

    dg::SparseTensor<dg::DVec> metric = g2d.metric();
    dg::DVec g_xx = metric.value(0,0), g_xy = metric.value(0,1), g_yy=metric.value(1,1);
    dg::DVec vol = dg::tensor::volume(metric);

    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    dg::blas1::axpby( 1., ones, -1., temp0, temp0);
    dg::assign( temp0, X);
    err = nc_put_var_double( ncid, defID, periodify(X, g3d_periodic).data());
    //err = nc_put_var_double( ncid, defID, X.data());
    dg::assign( vol, X);
    dg::assign( g_yy,Y);
    dg::blas1::pointwiseDot( Y, X, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g3d_periodic).data());
    //err = nc_put_var_double( ncid, volID, X.data());

    std::cout << "Construction successful!\n";

    //compute error in volume element (in conformal grid g^xx is the volume element)
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::assign( g_xx,  temp1);
    dg::blas1::pointwiseDot( temp1, temp1, temp1);
    dg::blas1::axpby( 1., temp1, -1., temp0, temp0);
    double error = sqrt( dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( temp1, w2d, temp1));
    std::cout<< "Rel Error in Determinant is "<<error<<"\n";

    //compute error in determinant vs volume form
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::assign( temp0, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g3d_periodic).data());
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( vol, w2d, vol));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    //compare g^xx to volume form
    dg::assign( g_xx, temp0);
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error=sqrt(dg::blas2::dot( temp0, w2d, temp0))/sqrt( dg::blas2::dot(vol, w2d, vol));
    std::cout << "Rel Error of volume form is "<<error<<"\n";

    std::cout << "TEST VOLUME IS:\n";
    dg::CartesianGrid2d g2dC( gp.R_0 -1.2*gp.a, gp.R_0 + 1.2*gp.a, -2.0*gp.a*gp.elongation, 1.2*gp.a*gp.elongation, 1, 5e3, 1e4, dg::PER, dg::PER);
    double psipmax = 0., psipmin = psi_0;
    auto iris = dg::compose( dg::Iris(  psipmin, psipmax), psip.f());
    dg::HVec vec  = dg::evaluate( iris, g2dC);
    dg::DVec cutter = dg::pullback( iris, g2d), cut_vol( cutter);
    dg::blas1::pointwiseDot(cutter, w2d, cut_vol);
    double volume = dg::blas1::dot( vol, cut_vol);
    dg::HVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = dg::blas1::dot( vec, g2d_weights);
    std::cout << "volumeXYP is "<< volume<<std::endl;
    std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    std::cout << "Note that the error might also come from the volume in RZP!\n";

    err = nc_close( ncid);
    return 0;
}
