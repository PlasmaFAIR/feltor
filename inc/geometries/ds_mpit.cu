#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <mpi.h>
#define DG_BENCHMARK
#undef DG_DEBUG
#include "dg/algorithm.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "ds.h"
#include "toroidal.h"

const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius

int main(int argc, char* argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz, mx[2], max_iter = 1e4;
    MPI_Comm comm;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    if(rank==0)std::cout << "# Test the parallel derivative DS in cylindrical coordinates for circular flux surfaces with DIR and NEU boundary conditions.\n";
    dg::mpi_init3d( dg::NEU, dg::NEU, dg::PER, n, Nx, Ny, Nz, comm);
    if( rank == 0)
    {
        std::cout <<"# You typed\n"
                  <<"n:  "<<n<<"\n"
                  <<"Nx: "<<Nx<<"\n"
                  <<"Ny: "<<Ny<<"\n"
                  <<"Nz: "<<Nz<<std::endl;
        std::cout <<"# Type mx (10) and my (10)\n";
        std::cin >> mx[0] >> mx[1];
        std::cout << "# You typed\n"
                  <<"mx: "<<mx[0]<<"\n"
                  <<"my: "<<mx[1]<<std::endl;
        std::cout << "# Create parallel Derivative!\n";
    }
    MPI_Bcast( mx, 2, MPI_INT, 0, MPI_COMM_WORLD);

    const dg::CylindricalMPIGrid3d g3d( R_0-a, R_0+a, -a, a, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::NEU, dg::NEU, dg::PER, comm);
    //create magnetic field
    const dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
    const dg::geo::CylindricalVectorLvl0 bhat( (dg::geo::BHatR)(mag), (dg::geo::BHatZ)(mag), (dg::geo::BHatP)(mag));
    //create Fieldaligned object and construct DS from it
    dg::geo::Fieldaligned<dg::aProductMPIGeometry3d,dg::MIDMatrix,dg::MDVec>  dsFA( bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1]);
    dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIDMatrix, dg::MDMatrix, dg::MDVec> ds( dsFA, dg::centered);
    ///##########################################################///
    dg::MDVec fun = dg::evaluate( dg::geo::TestFunctionDirNeu(mag), g3d);
    dg::MDVec derivative(fun);
    dg::MDVec divb = dg::evaluate( dg::geo::Divb(mag), g3d);
    dg::MDVec sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::TestFunctionDirNeu>(mag), g3d);
    dg::MDVec sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::TestFunctionDirNeu>(mag), g3d);
    dg::MDVec sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::TestFunctionDirNeu>(mag), g3d);
    dg::MDVec sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::TestFunctionDirNeu>(mag), g3d);
    dg::MDVec sol4 =dg::evaluate( dg::geo::OMDsDivDsFunction<dg::geo::TestFunctionDirNeu>(mag), g3d);
    std::vector<std::pair<std::string, std::array<const dg::MDVec*,2>>> names{
         {"forward",{&fun,&sol0}},          {"backward",{&fun,&sol0}},
         {"forward2",{&fun,&sol0}},         {"backward2",{&fun,&sol0}},
         {"centered",{&fun,&sol0}},         {"dss",{&fun,&sol1}},
         {"centered_bc_along",{&fun,&sol0}},{"dss_bc_along",{&fun,&sol1}},
         {"divForward",{&fun,&sol2}},       {"divBackward",{&fun,&sol2}},
         {"divCentered",{&fun,&sol2}},      {"divDirectForward",{&fun,&sol2}},
         {"divDirectBackward",{&fun,&sol2}},{"divDirectCentered",{&fun,&sol2}},
         {"forwardLap",{&fun,&sol3}},       {"backwardLap",{&fun,&sol3}},
         {"centeredLap",{&fun,&sol3}},      {"directLap",{&fun,&sol3}},
         {"directLap_bc_along",{&fun,&sol3}},
         {"invForwardLap",{&sol4,&fun}},    {"invBackwardLap",{&sol4,&fun}},
         {"invCenteredLap",{&sol4,&fun}}
    };
    if(rank==0)std::cout << "# TEST NEU Boundary conditions!\n";
    if(rank==0)std::cout << "# TEST ADJOINT derivatives do unfortunately not fulfill Neumann BC!\n";
    ///##########################################################///
    if(rank==0)std::cout <<"Neumann:\n";
    dg::MDVec vol3d = dg::create::volume( g3d);
    for( const auto& tuple :  names)
    {
        std::string name = std::get<0>(tuple);
        const dg::MDVec& function = *std::get<1>(tuple)[0];
        const dg::MDVec& solution = *std::get<1>(tuple)[1];
        callDS( ds, name, function, derivative, divb, max_iter,1e-8);
        double sol = dg::blas2::dot( vol3d, solution);
        dg::blas1::axpby( 1., solution, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    if(rank==0)std::cout << "# Reconstruct parallel derivative!\n";
    dsFA.construct( bhat, g3d, dg::DIR, dg::DIR, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1]);
    ds.construct( dsFA, dg::centered);
    if(rank==0)std::cout << "# TEST DIR Boundary conditions!\n";
    ///##########################################################///
    if(rank==0)std::cout << "Dirichlet: \n";
    for( const auto& tuple :  names)
    {
        std::string name = std::get<0>(tuple);
        const dg::MDVec& function = *std::get<1>(tuple)[0];
        const dg::MDVec& solution = *std::get<1>(tuple)[1];
        callDS( ds, name, function, derivative, divb, max_iter,1e-8);
        double sol = dg::blas2::dot( vol3d, solution);
        dg::blas1::axpby( 1., solution, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }

    ///##########################################################///
    if(rank==0)std::cout << "TEST FIELDALIGNED EVALUATION of a Gaussian\n";
    dg::Gaussian init0(R_0+0.5, 0, 0.2, 0.2, 1);
    dg::GaussianZ modulate(0., M_PI/3., 1);
    dg::MDVec aligned = dsFA.evaluate( init0, modulate, Nz/2, 2);
    ds( aligned, derivative);
    double norm = dg::blas2::dot(vol3d, derivative);
    if(rank==0)std::cout << "Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_t)\n";
    ///##########################################################///
    if(rank==0)std::cout << "# TEST STAGGERED GRID DERIVATIVE\n";
    dg::MDVec zMinus(fun), eMinus(fun), zPlus(fun), ePlus(fun);
    dg::MDVec funST(fun);
    dg::geo::Fieldaligned<dg::aProductMPIGeometry3d,dg::MIDMatrix,dg::MDVec>  dsFAST(
            bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1],
            g3d.hz()/2.);
    for( auto bc : {dg::NEU, dg::DIR})
    {
        if( bc == dg::DIR)
            if(rank==0)std::cout << "DirichletST:\n";
        if( bc == dg::NEU)
            if(rank==0)std::cout << "NeumannST:\n";
        dsFAST( dg::geo::zeroMinus, fun, zMinus);
        dsFAST( dg::geo::einsPlus,  fun, ePlus);
        dg::geo::assign_bc_along_field_1st( dsFAST, zMinus, ePlus, zMinus, ePlus,
            bc, {0,0});
        //dg::blas1::axpby( 0.5, zMinus, 0.5, ePlus, funST);
        dg::blas1::subroutine( []DG_DEVICE( double& funST, double zm, double ep,
                    double hp, double hm){
                funST = (hm*ep+hp*zm)/(hp+hm);
                }, funST, zMinus, ePlus, dsFAST.hp(), dsFAST.hm());
        dsFAST( dg::geo::zeroPlus, funST, zPlus);
        dsFAST( dg::geo::einsMinus, funST, eMinus);
        dg::geo::assign_bc_along_field_1st( dsFAST, eMinus, zPlus, eMinus, zPlus,
            bc, {0,0});
        dg::blas1::subroutine( []DG_DEVICE( double& df, double fm, double fp,
                    double hp, double hm){
                df = (fp-fm)/(hp+hm);
                }, derivative, eMinus, zPlus, dsFAST.hp(), dsFAST.hm());
        double sol = dg::blas2::dot( vol3d, sol0);
        dg::blas1::axpby( 1., sol0, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        std::string name = "forward";
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";

        // now try the adjoint direction (should be exactly the same result)
        dsFAST( dg::geo::zeroPlus, fun, zPlus);
        dsFAST( dg::geo::einsMinus, fun, eMinus);
        dg::geo::assign_bc_along_field_1st( dsFAST, eMinus, zPlus, eMinus, zPlus,
            bc, {0,0});
        //dg::blas1::axpby( 0.5, eMinus, 0.5, zPlus, funST);
        dg::blas1::subroutine( []DG_DEVICE( double& funST, double zm, double ep,
                    double hp, double hm){
                funST = (hm*ep+hp*zm)/(hp+hm);
                }, funST, eMinus, zPlus, dsFAST.hp(), dsFAST.hm());
        dsFAST( dg::geo::einsPlus, funST, ePlus);
        dsFAST( dg::geo::zeroMinus, funST, zMinus);
        dg::geo::assign_bc_along_field_1st( dsFAST, zMinus, ePlus, zMinus, ePlus,
            bc, {0,0});
        dg::blas1::subroutine( []DG_DEVICE( double& df, double fm, double fp,
                    double hp, double hm){
                df = (fp-fm)/(hp+hm);
                }, derivative, zMinus, ePlus, dsFAST.hp(), dsFAST.hm());
        dg::blas1::axpby( 1., sol0, -1., derivative);
        norm = dg::blas2::dot( derivative, vol3d, derivative);
        name = "backward";
        if(rank==0)std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    MPI_Finalize();
    return 0;
}
