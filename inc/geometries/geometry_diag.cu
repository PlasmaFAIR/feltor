#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <functional>
#include <sstream>
#include <ctime>
#include <cmath>

#include "dg/algorithm.h"
#include "dg/file/file.h"

#include "make_field.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "curvilinearX.h"
#include "separatrix_orthogonal.h"
#include "average.h"


// The purpose of this program is to diagnose geometry.json files with
// as little effort as possible. This program should also remain
// independent of any specific code and therefore does not test or output
// any initialization related functions that require specific parameters in
// the input file
// We currently just
// - write magnetic functions into file
// - compute Flux - surface averages and write into file
//

int main( int argc, char* argv[])
{
    dg::file::WrappedJsonValue js( dg::file::error::is_warning);
    std::string inputfile = argc==1 ? "geometry_diag.json" : argv[1];
    dg::file::file2Json( inputfile, js.asJson(),
            dg::file::comments::are_discarded);

    std::string geometry_params = js["magnetic_field"]["input"].asString();
    if( geometry_params == "file")
    {
        std::string path = js["magnetic_field"]["file"].asString();
        dg::file::file2Json( path, js.asJson()["magnetic_field"]["file"],
                dg::file::comments::are_discarded);
    }
    //Test coefficients
    dg::geo::CylindricalFunctor wall, transition, sheath;
    dg::geo::TokamakMagneticField mag = dg::geo::createMagneticField(
            js["magnetic_field"][geometry_params]);
    dg::geo::TokamakMagneticField mod_mag =
        dg::geo::createModifiedField(js["magnetic_field"][geometry_params],
                js["boundary"]["wall"], wall, transition);
    unsigned n = js["grid"].get("n",3).asUInt();
    unsigned Nx = js["grid"].get("Nx",100).asUInt();
    unsigned Ny = js["grid"].get("Ny",100).asUInt();
    unsigned Nz = js["grid"].get("Nz", 1).asUInt();
    double boxscaleRm = js["grid"]["scaleR"].get(0u, 1.1).asDouble();
    double boxscaleRp = js["grid"]["scaleR"].get(1u, 1.1).asDouble();
    double boxscaleZm = js["grid"]["scaleZ"].get(0u, 1.2).asDouble();
    double boxscaleZp = js["grid"]["scaleZ"].get(1u, 1.1).asDouble();
    double Rmin=mag.R0()-boxscaleRm*mag.params().a();
    double Zmin=-boxscaleZm*mag.params().a();
    double Rmax=mag.R0()+boxscaleRp*mag.params().a();
    double Zmax=boxscaleZp*mag.params().a();
    dg::Grid2d sheath_walls( Rmin, Rmax, Zmin, Zmax, 1,1,1);
    //std::string sheath_bc = js["boundary"]["sheath"].get("type", "none").asString();
    //if( sheath_bc != "none")
    dg::geo::createSheathRegion( js["boundary"]["sheath"],
            mag, wall, sheath_walls, sheath);

    dg::geo::description mag_description = mag.params().getDescription();

    dg::Grid2d grid2d(Rmin,Rmax,Zmin,Zmax, n,Nx,Ny);
    dg::DVec psipog2d   = dg::evaluate( mag.psip(), grid2d);
    double psipO = dg::blas1::reduce( psipog2d, 0., thrust::minimum<double>());
    double psipmax = dg::blas1::reduce( psipog2d, 0., thrust::maximum<double>());
    if( mag_description == dg::geo::description::standardX ||
        mag_description == dg::geo::description::standardO ||
        mag_description == dg::geo::description::square ||
        mag_description == dg::geo::description::doubleX
        )
    {
        //Find O-point
        double RO = mag.R0(), ZO = 0.;
        int point = dg::geo::findOpoint( mag.get_psip(), RO, ZO);
        psipO = mag.psip()( RO, ZO);
        std::cout << "O-point found at "<<RO<<" "<<ZO<<" with Psip "<<psipO<<std::endl;
        if( point == 1 )
            std::cout << " (minimum)"<<std::endl;
        if( point == 2 )
            std::cout << " (maximum)"<<std::endl;
        double psip0 = mag.psip()(mag.R0(), 0);
        std::cout << "psip( R_0, 0) = "<<psip0<<"\n";
        double fx_0 = 1./8.;
        psipmax = -fx_0/(1.-fx_0)*psipO;
    }

    std::vector<std::tuple<std::string, dg::HVec, std::string> > map1d;
    //Generate list of functions to evaluate
    std::vector< std::tuple<std::string, std::string, dg::geo::CylindricalFunctor >> map{
        {"Psip", "Flux function", mag.psip()},
        {"PsipR", "Flux function derivative in R", mag.psipR()},
        {"PsipZ", "Flux function derivative in Z", mag.psipZ()},
        {"PsipRR", "Flux function derivative in RR", mag.psipRR()},
        {"PsipRZ", "Flux function derivative in RZ", mag.psipRZ()},
        {"PsipZZ", "Flux function derivative in ZZ", mag.psipZZ()},
        {"Ipol", "Poloidal current", mag.ipol()},
        {"IpolR", "Poloidal current derivative in R", mag.ipolR()},
        {"IpolZ", "Poloidal current derivative in Z", mag.ipolZ()},
        {"Rho_p", "Normalized Poloidal flux label", dg::geo::RhoP(mag)},
        {"Bmodule", "Magnetic field strength", dg::geo::Bmodule(mag)},
        {"InvB", "Inverse of Bmodule", dg::geo::InvB(mag)},
        {"LnB", "Natural logarithm of Bmodule", dg::geo::LnB(mag)},
        {"GradLnB", "The parallel derivative of LnB", dg::geo::GradLnB(mag)},
        {"Divb", "The divergence of the magnetic unit vector", dg::geo::Divb(mag)},
        {"B_R", "Derivative of Bmodule in R", dg::geo::BR(mag)},
        {"B_Z", "Derivative of Bmodule in Z", dg::geo::BZ(mag)},
        {"CurvatureNablaBR",  "R-component of the (toroidal) Nabla B curvature vector", dg::geo::CurvatureNablaBR(mag,+1)},
        {"CurvatureNablaBZ",  "Z-component of the (toroidal) Nabla B curvature vector", dg::geo::CurvatureNablaBZ(mag,+1)},
        {"CurvatureKappaR",   "R-component of the (toroidal) Kappa B curvature vector", dg::geo::CurvatureKappaR(mag,+1)},
        {"CurvatureKappaZ",   "Z-component of the (toroidal) Kappa B curvature vector", dg::geo::CurvatureKappaZ(mag,+1)},
        {"DivCurvatureKappa", "Divergence of the (toroidal) Kappa B curvature vector", dg::geo::DivCurvatureKappa(mag,+1)},
        {"DivCurvatureNablaB","Divergence of the (toroidal) Nabla B curvature vector", dg::geo::DivCurvatureNablaB(mag,+1)},
        {"TrueCurvatureNablaBR", "R-component of the (true) Nabla B curvature vector", dg::geo::TrueCurvatureNablaBR(mag)},
        {"TrueCurvatureNablaBZ", "Z-component of the (true) Nabla B curvature vector", dg::geo::TrueCurvatureNablaBZ(mag)},
        {"TrueCurvatureNablaBP", "Contravariant Phi-component of the (true) Nabla B curvature vector", dg::geo::TrueCurvatureNablaBP(mag)},
        {"TrueCurvatureKappaR", "R-component of the (true) Kappa B curvature vector", dg::geo::TrueCurvatureKappaR(mag)},
        {"TrueCurvatureKappaZ", "Z-component of the (true) Kappa B curvature vector", dg::geo::TrueCurvatureKappaZ(mag)},
        {"TrueCurvatureKappaP", "Contravariant Phi-component of the (true) Kappa B curvature vector", dg::geo::TrueCurvatureKappaP(mag)},
        {"TrueDivCurvatureKappa", "Divergence of the (true) Kappa B curvature vector", dg::geo::TrueDivCurvatureKappa(mag)},
        {"TrueDivCurvatureNablaB","Divergence of the (true) Nabla B curvature vector",  dg::geo::TrueDivCurvatureNablaB(mag)},
        {"BFieldR", "R-component of the magnetic field vector", dg::geo::BFieldR(mag)},
        {"BFieldZ", "Z-component of the magnetic field vector", dg::geo::BFieldZ(mag)},
        {"BFieldP", "Contravariant Phi-component of the magnetic field vector", dg::geo::BFieldP(mag)},
        {"BHatR", "R-component of the magnetic field unit vector", dg::geo::BHatR(mag)},
        {"BHatZ", "Z-component of the magnetic field unit vector", dg::geo::BHatZ(mag)},
        {"BHatP", "Contravariant Phi-component of the magnetic field unit vector", dg::geo::BHatP(mag)},
        {"GradBHatR", "Parallel derivative of BHatR", dg::geo::BHatR(mag)},
        {"GradBHatZ", "Parallel derivative of BHatZ", dg::geo::BHatZ(mag)},
        {"GradBHatP", "Parallel derivative of BHatP", dg::geo::BHatP(mag)},
        {"NormGradPsip", "Norm of gradient of Psip", dg::geo::SquareNorm( dg::geo::createGradPsip(mag), dg::geo::createGradPsip(mag))},
        {"CurvatureNablaBGradPsip", "(Toroidal) Nabla B curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createCurvatureNablaB(mag, +1), dg::geo::createGradPsip(mag))},
        {"CurvatureKappaGradPsip", "(Toroidal) Kappa curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createCurvatureKappa(mag, +1), dg::geo::createGradPsip(mag))},
        {"TrueCurvatureNablaBGradPsip", "True Nabla B curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createTrueCurvatureNablaB(mag), dg::geo::createGradPsip(mag))},
        {"TrueCurvatureKappaGradPsip", "True Kappa curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createTrueCurvatureKappa(mag), dg::geo::createGradPsip(mag))},
        /////////////////////////////////////
        {"WallDistance", "Distance to closest wall", dg::geo::CylindricalFunctor( dg::WallDistance( sheath_walls)) },
        {"WallFieldlineAnglePDistance", "Distance to wall along fieldline",
            dg::geo::WallFieldlineDistance( dg::geo::createBHat(mod_mag),
                    sheath_walls, 4*2.*M_PI, 1e-6, "phi") },
        {"WallFieldlineAngleMDistance", "Distance to wall along fieldline",
            dg::geo::WallFieldlineDistance( dg::geo::createBHat(mod_mag),
                    sheath_walls, -4*2.*M_PI, 1e-6, "phi") },
        {"WallFieldlineSPDistance", "Distance to wall along fieldline",
            dg::geo::WallFieldlineDistance( dg::geo::createBHat(mod_mag),
                    sheath_walls, 4*2.*M_PI, 1e-6, "s") },
        {"WallFieldlineSMDistance", "Distance to wall along fieldline",
            dg::geo::WallFieldlineDistance( dg::geo::createBHat(mod_mag),
                    sheath_walls, -4*2.*M_PI, 1e-6, "s") },
        {"Sheath", "Sheath region", sheath},
        {"SheathDirection", "Direction of magnetic field relative to sheath", dg::geo::WallDirection(mag, sheath_walls) },
        {"SheathCoordinate", "Coordinate from -1 to 1 of magnetic field relative to sheath", dg::geo::WallFieldlineCoordinate( dg::geo::createBHat( mod_mag), sheath_walls, 4*2.*M_PI, 1e-6, "s")},
        //////////////////////////////////
        {"Iris", "A flux aligned Iris", dg::compose( dg::Iris( 0.5, 0.7), dg::geo::RhoP(mag))},
        {"Pupil", "A flux aligned Pupil", dg::compose( dg::Pupil(0.7), dg::geo::RhoP(mag)) },
        {"PsiLimiter", "A flux aligned Heaviside", dg::compose( dg::Heaviside( 1.03), dg::geo::RhoP(mag) )},
        {"Wall", "Penalization region that acts as the wall", wall },
        {"MagneticTransition", "The region where the magnetic field is modified", transition},
        {"Delta", "A flux aligned Gaussian peak", dg::compose( dg::GaussianX( psipO*0.2, 0.1, 1./(sqrt(2.*M_PI)*0.1)), mag.psip())},
        ////
        { "Hoo", "The novel h02 factor", dg::geo::Hoo( mag) }
    };

    ///////////TEST CURVILINEAR GRID TO COMPUTE FSA QUANTITIES
    unsigned npsi = 3;
    //set number of psivalues (NPsi % 8 == 0)
    unsigned Npsi = js["grid"].get("Npsi", 32).asUInt();
    unsigned Neta = js["grid"].get("Neta", 640).asUInt();
    /// -------  Elements for fsa on X-point grid ----------------
    std::unique_ptr<dg::geo::CurvilinearGridX2d> gX2d;
    if( mag_description == dg::geo::description::standardX)
    {
        std::cout << "Generate X-point flux-aligned grid ... \n";
        double RX = mag.R0()-1.1*mag.params().triangularity()*mag.params().a();
        double ZX = -1.1*mag.params().elongation()*mag.params().a();
        dg::geo::findXpoint( mag.get_psip(), RX, ZX);
        double psipX = mag.psip()(RX, ZX);
        std::cout << "X-point found at "<<RX<<" "<<ZX<<" with Psip = "<<psipX<<std::endl;
        if( fabs(psipX ) > 1e-10)
        {
            std::cerr << " Psip at X-point is not zero. Unable to construct grid\n";
            return -1;
        }
        dg::geo::CylindricalSymmTensorLvl1 monitor_chi = dg::geo::make_Xconst_monitor( mag.get_psip(), RX, ZX) ;
        dg::geo::SeparatrixOrthogonal generator(mag.get_psip(), monitor_chi, psipO, RX, ZX, mag.R0(), 0, 0, true);
        double fx_0 = 1./8.;
        psipmax = -fx_0/(1.-fx_0)*psipO;
        std::cout << "psi 1 is          "<<psipmax<<"\n";
        gX2d = std::make_unique<dg::geo::CurvilinearGridX2d>(generator, fx_0, 0., npsi, Npsi, Neta, dg::DIR, dg::NEU);
        std::cout << "DONE! \n";
        dg::Average<dg::HVec > avg_eta( gX2d->grid(), dg::coo2d::y);
        std::vector<dg::HVec> coordsX = gX2d->map();
        dg::SparseTensor<dg::HVec> metricX = gX2d->metric();
        dg::HVec volX2d = dg::tensor::volume2d( metricX);
        dg::blas1::pointwiseDot( coordsX[0], volX2d, volX2d); //R\sqrt{g}
        const double f0 = (gX2d->x1()-gX2d->x0())/ ( psipmax - psipO);
        dg::HVec dvdpsip;
        avg_eta( volX2d, dvdpsip, false);
        dg::blas1::scal( dvdpsip, 4.*M_PI*M_PI*f0);
        dg::Grid1d gX1d(psipO<psipmax ? psipO : psipmax,
            psipO<psipmax ? psipmax : psipO, npsi ,Npsi,dg::DIR_NEU); //inner value is always zero
        dg::HVec X_psi_vol = dg::integrate( dvdpsip, gX1d);
        map1d.emplace_back( "dvdpsip", dvdpsip,
            "Derivative of flux volume with respect to flux label psi");
        map1d.emplace_back( "psi_vol", X_psi_vol,
            "Flux volume on X-point grid");

        //NOTE: VOLUME is WITHIN cells while AREA is ON gridpoints
        dg::HVec gradZetaX = metricX.value(0,0), X_psi_area;
        dg::blas1::transform( gradZetaX, gradZetaX, dg::SQRT<double>());
        dg::blas1::pointwiseDot( volX2d, gradZetaX, gradZetaX); //R\sqrt{g}|\nabla\zeta|
        avg_eta( gradZetaX, X_psi_area, false);
        dg::blas1::scal( X_psi_area, 4.*M_PI*M_PI);
        map1d.emplace_back( "psi_area", X_psi_area,
            "Flux area on X-point grid");
        std::cout << "Total volume within separatrix is "<< dg::interpolate( dg::xspace, X_psi_vol, 0., gX1d)<<std::endl;

        //Compute FSA of cylindrical functions
        dg::HVec transferH, transferH1d;
        for( auto tp : map)
        {
            if( std::get<0>(tp).find("Wall") != std::string::npos)
                continue;
            if( std::get<0>(tp).find("Sheath") != std::string::npos)
                continue;
            transferH = dg::pullback( std::get<2>(tp), *gX2d);
            dg::blas1::pointwiseDot( volX2d, transferH, transferH);
            avg_eta( transferH, transferH1d, false);
            dg::blas1::scal( transferH1d, 4*M_PI*M_PI*f0); //
            dg::blas1::pointwiseDivide( transferH1d, dvdpsip, transferH1d );
            map1d.emplace_back( std::get<0>(tp)+"_fsa", transferH1d,
                std::get<1>(tp)+" (Flux surface average)");
            dg::blas1::pointwiseDot( transferH1d, dvdpsip, transferH1d );
            transferH1d = dg::integrate( transferH1d, gX1d);
            map1d.emplace_back( std::get<0>(tp)+"_ifs", transferH1d,
                std::get<1>(tp)+" (Flux surface integral)");

        }
    }
    /// --------- More flux labels --------------------------------
    dg::Grid1d grid1d(psipO<psipmax ? psipO : psipmax,
            psipO<psipmax ? psipmax : psipO, npsi ,Npsi,dg::DIR_NEU); //inner value is always zero
    if( mag_description == dg::geo::description::standardX ||
        mag_description == dg::geo::description::standardO ||
        mag_description == dg::geo::description::square ||
        mag_description == dg::geo::description::doubleX
        )
    {
        dg::HVec rho = dg::evaluate( dg::cooX1d, grid1d);
        dg::blas1::axpby( -1./psipO, rho, +1., 1., rho); //transform psi to rho
        map1d.emplace_back("rho", rho,
            "Alternative flux label rho = -psi/psimin + 1");
        dg::blas1::transform( rho, rho, dg::SQRT<double>());
        map1d.emplace_back("rho_p", rho,
            "Alternative flux label rho_p = Sqrt[-psi/psimin + 1]");
        dg::geo::SafetyFactor qprof( mag);
        dg::HVec psi_vals = dg::evaluate( dg::cooX1d, grid1d);
        // we need to avoid calling SafetyFactor outside closed fieldlines
        dg::blas1::subroutine( [psipO]( double& psi){
               if( (psipO < 0 && psi > 0) || (psipO>0 && psi <0))
                   psi = psipO/2.; // just use a random value
            }, psi_vals);
        dg::HVec qprofile( psi_vals);
        dg::blas1::evaluate( qprofile, dg::equals(), qprof, psi_vals);
        map1d.emplace_back("q-profile", qprofile,
            "q-profile (Safety factor) using direct integration");
        dg::HVec psit = dg::integrate( qprofile, grid1d);
        map1d.emplace_back("psit1d", psit,
            "Toroidal flux label psi_t integrated  on grid1d using direct q");
        //we need to avoid integrating outside closed fieldlines
        dg::Grid1d g1d_fine(psipO<0. ? psipO : 0.,
                psipO<0. ? 0. : psipO, npsi, Npsi,dg::NEU);
        qprofile = dg::evaluate( qprof, g1d_fine);
        dg::HVec w1d = dg::create::weights( g1d_fine);
        double psit_tot = dg::blas1::dot( w1d, qprofile);
        //std::cout << "psit tot "<<psit_tot<<"\n";
        dg::blas1::scal ( psit, 1./psit_tot);
        dg::blas1::transform( psit, psit, dg::SQRT<double>());
        map1d.emplace_back("rho_t", psit,
            "Toroidal flux label rho_t = sqrt( psit/psit_tot) evaluated on grid1d");
    }

    /////////////////////////////set up netcdf/////////////////////////////////////
    std::cout << "CREATING/OPENING FILE ... \n";
    dg::file::NC_Error_Handle err;
    int ncid;
    std::string newfilename = argc<3 ? "geometry_diag.nc" : argv[2];
    err = nc_create( newfilename.c_str(), NC_NETCDF4|NC_CLOBBER, &ncid);
    /// Set global attributes
    std::map<std::string, std::string> att;
    att["title"] = "Output file of feltor/inc/geometries/geometry_diag.cu";
    att["Conventions"] = "CF-1.7";
    ///Get local time and begin file history
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);

    std::ostringstream oss;
    ///time string  + program-name + args
    oss << std::put_time(&tm, "%Y-%m-%d %H:%M:%S");
    for( int i=0; i<argc; i++) oss << " "<<argv[i];
    att["history"] = oss.str();
    att["comment"] = "Find more info in feltor/src/feltor.tex";
    att["source"] = "FELTOR";
    att["references"] = "https://github.com/feltor-dev/feltor";
    std::string input = js.asJson().toStyledString();
    att["inputfile"] = input;
    for( auto pair : att)
        err = nc_put_att_text( ncid, NC_GLOBAL,
            pair.first.data(), pair.second.size(), pair.second.data());

    int dim1d_ids[1], dim2d_ids[2], dim3d_ids[3] ;
    if( mag_description == dg::geo::description::standardX)
    {
        int dim_idsX[2] = {0,0};
        err = dg::file::define_dimensions( ncid, dim_idsX, gX2d->grid(), {"eta", "zeta"} );
        std::string long_name = "Flux surface label";
        err = nc_put_att_text( ncid, dim_idsX[0], "long_name",
            long_name.size(), long_name.data());
        long_name = "Flux angle";
        err = nc_put_att_text( ncid, dim_idsX[1], "long_name",
            long_name.size(), long_name.data());
        int xccID, yccID;
        err = nc_def_var( ncid, "xcc", NC_DOUBLE, 2, dim_idsX, &xccID);
        err = nc_def_var( ncid, "ycc", NC_DOUBLE, 2, dim_idsX, &yccID);
        long_name="Cartesian x-coordinate";
        err = nc_put_att_text( ncid, xccID, "long_name",
            long_name.size(), long_name.data());
        long_name="Cartesian y-coordinate";
        err = nc_put_att_text( ncid, yccID, "long_name",
            long_name.size(), long_name.data());
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, xccID, gX2d->map()[0].data());
        err = nc_put_var_double( ncid, yccID, gX2d->map()[1].data());
        err = nc_redef(ncid);
        dim1d_ids[0] = dim_idsX[1];
    }
    else
    {
        err = dg::file::define_dimension( ncid, &dim1d_ids[0], grid1d, "zeta");
        std::string psi_long_name = "Flux surface label";
        err = nc_put_att_text( ncid, dim1d_ids[0], "long_name",
            psi_long_name.size(), psi_long_name.data());
    }
    dg::CylindricalGrid3d grid3d(Rmin,Rmax,Zmin,Zmax, 0, 2.*M_PI, n,Nx,Ny,Nz);
    dg::RealCylindricalGrid3d<float> fgrid3d(Rmin,Rmax,Zmin,Zmax, 0, 2.*M_PI, n,Nx,Ny,Nz);

    err = dg::file::define_dimensions( ncid, &dim3d_ids[0], fgrid3d);
    dim2d_ids[0] = dim3d_ids[1], dim2d_ids[1] = dim3d_ids[2];

    //write 1d vectors
    std::cout << "WRTING 1D FIELDS ... \n";
    for( auto tp : map1d)
    {
        int vid;
        err = nc_def_var( ncid, std::get<0>(tp).data(), NC_DOUBLE, 1,
            &dim1d_ids[0], &vid);
        err = nc_put_att_text( ncid, vid, "long_name",
            std::get<2>(tp).size(), std::get<2>(tp).data());
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, vid, std::get<1>(tp).data());
        err = nc_redef(ncid);
    }
    //write 2d vectors
    //allocate mem for visual
    dg::HVec hvisual = dg::evaluate( dg::zero, grid2d);
    dg::HVec hvisual3d = dg::evaluate( dg::zero, grid3d);
    dg::fHVec fvisual, fvisual3d;
    std::cout << "WRTING 2D/3D CYLINDRICAL FIELDS ... \n";
    for(auto tp : map)
    {
        int vectorID, vectorID3d;
        err = nc_def_var( ncid, std::get<0>(tp).data(), NC_FLOAT, 2,
            &dim2d_ids[0], &vectorID);
        err = nc_def_var( ncid, (std::get<0>(tp)+"3d").data(), NC_FLOAT, 3,
            &dim3d_ids[0], &vectorID3d);
        err = nc_put_att_text( ncid, vectorID, "long_name",
            std::get<1>(tp).size(), std::get<1>(tp).data());
        err = nc_put_att_text( ncid, vectorID3d, "long_name",
            std::get<1>(tp).size(), std::get<1>(tp).data());
        std::string coordinates = "zc yc xc";
        err = nc_put_att_text( ncid, vectorID3d, "coordinates", coordinates.size(), coordinates.data());
        err = nc_enddef( ncid);
        dg::Timer t;
        t.tic();
        hvisual = dg::evaluate( std::get<2>(tp), grid2d);
        t.toc();
        if((    std::get<0>(tp).find("Wall") != std::string::npos)
            ||( std::get<0>(tp).find("Sheath") != std::string::npos))
            std::cout<< std::get<0>(tp) << " took "<<t.diff()<<"s\n";
        dg::extend_line( grid2d.size(), grid3d.Nz(), hvisual, hvisual3d);
        dg::assign( hvisual, fvisual);
        dg::assign( hvisual3d, fvisual3d);
        err = nc_put_var_float( ncid, vectorID, fvisual.data());
        err = nc_put_var_float( ncid, vectorID3d, fvisual3d.data());
        err = nc_redef(ncid);
    }
    std::cout << "WRTING 3D FIELDS ... \n";
    //compute & write 3d vectors
    std::vector< std::tuple<std::string, std::string, std::function< double(double,double,double)> > > map3d{
        {"BR", "R-component of the magnetic field vector (3d version of BFieldR)",
            dg::geo::BFieldR(mag)},
        {"BZ", "Z-component of the magnetic field vector (3d version of BFieldZ)",
            dg::geo::BFieldZ(mag)},
        {"BP", "Contravariant Phi-component of the magnetic field vector (3d version of BFieldP)",
            dg::geo::BFieldP(mag)},
        {"xc", "x-coordinate in Cartesian coordinate system", dg::cooRZP2X},
        {"yc", "y-coordinate in Cartesian coordinate system", dg::cooRZP2Y},
        {"zc", "z-coordinate in Cartesian coordinate system", dg::cooRZP2Z}
    };
    for( auto tp : map3d)
    {
        int vectorID;
        err = nc_def_var( ncid, std::get<0>(tp).data(), NC_FLOAT, 3,
            &dim3d_ids[0], &vectorID);
        err = nc_put_att_text( ncid, vectorID, "long_name",
            std::get<1>(tp).size(), std::get<1>(tp).data());
        if( std::get<1>(tp) != "xc" && std::get<1>(tp) != "yc" &&std::get<1>(tp) != "zc")
        {
            std::string coordinates = "zc yc xc";
            err = nc_put_att_text( ncid, vectorID, "coordinates", coordinates.size(), coordinates.data());
        }
        err = nc_enddef( ncid);
        hvisual3d = dg::evaluate( std::get<2>(tp), grid3d);
        dg::assign( hvisual3d, fvisual3d);
        err = nc_put_var_float( ncid, vectorID, fvisual3d.data());
        err = nc_redef(ncid);
    }
    //////////////////////////////Finalize////////////////////////////////////
    err = nc_close(ncid);
    return 0;
}
