#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#define DG_BENCHMARK
#include "dg/algorithm.h"
#include "ds.h"
#include "guenther.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "ds_generator.h"

const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius

int main( )
{
    std::cout << "# Test the parallel derivative DS in cylindrical coordinates for the guenther flux surfaces. Fieldlines do not cross boundaries.\n";
    std::cout << "# Type n (3), Nx(20), Ny(20), Nz(20)\n";
    unsigned n, Nx, Ny, Nz, mx, my, max_iter = 1e4;
    std::string method = "cubic";
    std::cin >> n>> Nx>>Ny>>Nz;
    std::cout <<"# You typed\n"
              <<"n:  "<<n<<"\n"
              <<"Nx: "<<Nx<<"\n"
              <<"Ny: "<<Ny<<"\n"
              <<"Nz: "<<Nz<<std::endl;
    std::cout << "# Type mx (10) and my (10)\n";
    std::cin >> mx>> my;
    std::cout << "# You typed\n"
              <<"mx: "<<mx<<"\n"
              <<"my: "<<my<<std::endl;
    std::cout << "# Type method (dg, nearest, linear, cubic) \n";
    std::cin >> method;
    method.erase( std::remove( method.begin(), method.end(), '"'), method.end());
    std::cout << "# You typed\n"
              <<"method: "<< method<<std::endl;
    std::cout << "# Create parallel Derivative!\n";
    ////////////////////////////////initialze fields /////////////////////
    const dg::CylindricalGrid3d g3d( R_0 - a, R_0+a, -a, a, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::NEU, dg::NEU, dg::PER);
    const dg::geo::TokamakMagneticField mag = dg::geo::createGuentherField(R_0, I_0);
    dg::geo::DS<dg::aProductGeometry3d, dg::IDMatrix, dg::DMatrix, dg::DVec> ds(
        mag, g3d, dg::NEU, dg::NEU, dg::geo::FullLimiter(),
        1e-8, mx, my, -1, method);

    ///##########################################################///
    const dg::DVec fun = dg::evaluate( dg::geo::TestFunctionPsi2(mag), g3d);
    dg::DVec derivative(fun);
    dg::DVec sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::TestFunctionPsi2>(mag), g3d);
    dg::DVec sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::TestFunctionPsi2>(mag), g3d);
    dg::DVec sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::TestFunctionPsi2>(mag), g3d);
    dg::DVec sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::TestFunctionPsi2>(mag), g3d);
    dg::DVec sol4 = dg::evaluate( dg::geo::OMDsDivDsFunction<dg::geo::TestFunctionPsi2>(mag), g3d);
    std::vector<std::pair<std::string, std::array<const dg::DVec*,2>>> names{
         {"forward",{&fun,&sol0}},          {"backward",{&fun,&sol0}},
         {"forward2",{&fun,&sol0}},         {"backward2",{&fun,&sol0}},
         {"centered",{&fun,&sol0}},         {"dss",{&fun,&sol1}},
         {"centered_bc_along",{&fun,&sol0}},{"dss_bc_along",{&fun,&sol1}},
         {"divForward",{&fun,&sol2}},       {"divBackward",{&fun,&sol2}},
         {"divCentered",{&fun,&sol2}},      {"directLap",{&fun,&sol3}},
         {"invCenteredLap",{&sol4,&fun}}
    };

    ///##########################################################///
    std::cout << "# TEST Guenther (No Boundary conditions)!\n";
    std::cout <<"Guenther : #rel_Error rel_Volume_integral(should be zero for div and Lap)\n";
    const dg::DVec vol3d = dg::create::volume( g3d);
    for( const auto& tuple :  names)
    {
        std::string name = std::get<0>(tuple);
        const dg::DVec& function = *std::get<1>(tuple)[0];
        const dg::DVec& solution = *std::get<1>(tuple)[1];
        callDS( ds, name, function, derivative, max_iter,1e-8);
        double sol = dg::blas2::dot( vol3d, solution);
        double vol = dg::blas1::dot( vol3d, derivative)/sqrt( dg::blas2::dot( vol3d, function)); // using function in denominator makes entries comparable
        dg::blas1::axpby( 1., solution, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
                  <<" "<<sqrt(norm/sol)<<std::endl
                  <<"    "<<name+"_vol:"<<std::setw(30-name.size())
                  <<" "<<vol<<"\n";
    }
    ///##########################################################///
    std::cout << "# TEST TOTAL VARIATION DIMINISHING\n";
    ds.fieldaligned()(dg::geo::zeroPlus, fun, derivative);
    double mass_before = dg::blas1::dot( vol3d, fun);
    std::cout << "# mass before: "<<mass_before<<"\n";
    double mass_after = dg::blas1::dot( ds.fieldaligned().sqrtGp(), derivative);
    std::cout << "# mass after   "<<mass_after<<"\n";
    std::cout << "# Difference   "<<fabs(mass_before-mass_after)/mass_before<<"\n";
    mass_before = sqrt(dg::blas2::dot( vol3d, fun));
    std::cout << "# l2 norm before: "<<mass_before<<"\n";
    mass_after = sqrt(dg::blas2::dot( ds.fieldaligned().sqrtGp(), derivative));
    std::cout << "# l2 norm after   "<<mass_after<<"\n";
    std::cout << "# Difference   "<<fabs(mass_before-mass_after)/mass_before<<"\n";

    dg::geo::DSPGenerator generator( mag, g3d.x0(), g3d.x1(), g3d.y0(), g3d.y1(), g3d.hz());
    dg::geo::CurvilinearProductGrid3d g3dP( generator, g3d.n(), g3d.Nx(),
        g3d.Ny(), g3d.Nz(), g3d.bcx(), g3d.bcy(), g3d.bcz());
    dg::DVec vol3dP = dg::create::volume( g3dP);

    std::cout << "# Volume on original    grid: "<<dg::blas1::dot( 1., vol3d)<<"\n";
    std::cout << "# Volume on curvilinear grid: "<<dg::blas1::dot( 1., vol3dP)<<"\n";
    std::cout << "# Difference                : "<<dg::blas1::dot( 1., vol3d) - dg::blas1::dot( 1., vol3dP)<<"\n";
    dg::Elliptic<dg::aProductGeometry3d, dg::DMatrix, dg::DVec> elliptic(g3d,
        dg::normed);
    dg::DVec variation(fun);
    elliptic.variation( fun, variation);
    dg::blas1::transform( variation, variation, []DG_DEVICE( double var){ return var/sqrt(var);});
    double var_before = dg::blas1::dot( vol3d, variation);
    std::cout << "# variation before: "<<var_before<<"\n";
    elliptic.construct(g3dP, dg::normed);
    elliptic.variation( derivative, variation);
    dg::blas1::transform( variation, variation, []DG_DEVICE( double var){ return var/sqrt(var);});
    double var_after = dg::blas1::dot( vol3d, variation);
    std::cout << "# variation after   "<<var_after<<"\n";
    std::cout << "# Difference (Af-Be)"<<var_after-var_before<<"\n";
    ///##########################################################///
    std::cout << "# TEST STAGGERED GRID DERIVATIVE\n";
    dg::DVec zMinus(fun), eMinus(fun), zPlus(fun), ePlus(fun);
    dg::DVec funST(fun);
    dg::geo::Fieldaligned<dg::aProductGeometry3d,dg::IDMatrix,dg::DVec>  dsFAST(
            mag, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx, my,
            g3d.hz()/2., method);
    dsFAST( dg::geo::zeroMinus, fun, zMinus);
    dsFAST( dg::geo::einsPlus,  fun, ePlus);
    dg::geo::ds_slope( dsFAST, 1., zMinus, ePlus, 0., funST);
    dsFAST( dg::geo::zeroPlus, funST, zPlus);
    dsFAST( dg::geo::einsMinus, funST, eMinus);
    dg::geo::ds_average( dsFAST, 1., eMinus, zPlus, 0., derivative);

    double sol = dg::blas2::dot( vol3d, sol0);
    double vol = dg::blas1::dot( vol3d, derivative)/sqrt( dg::blas2::dot( vol3d, fun));
    dg::blas1::axpby( 1., sol0, -1., derivative);
    double norm = dg::blas2::dot( derivative, vol3d, derivative);
    std::string name  = "centeredST";
    std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
              <<" "<<sqrt(norm/sol)<<"\n"
              <<"    "<<name+"_vol:"<<std::setw(30-name.size())
              <<" "<<vol<<"\n";

    ds.fieldaligned()(dg::geo::einsPlus, fun, ePlus);
    ds.fieldaligned()(dg::geo::einsMinus, fun, eMinus);
    dg::blas1::pointwiseDot ( 1./2./dsFAST.deltaPhi(), dsFAST.bphiM(),
            fun, -1./2./dsFAST.deltaPhi(), dsFAST.bphiM(),
            eMinus, 0., eMinus);
    dg::blas1::pointwiseDot( 1./2./dsFAST.deltaPhi(), ePlus,
            dsFAST.bphiP(), -1./2./dsFAST.deltaPhi(), fun,
            dsFAST.bphiP(), 0., ePlus);
    dg::geo::ds_divCentered( dsFAST, 1., eMinus, ePlus, 0., derivative);
    sol = dg::blas2::dot( vol3d, sol3);
    vol = dg::blas1::dot( vol3d, derivative)/sqrt( dg::blas2::dot( vol3d, fun));
    dg::blas1::axpby( 1., sol3, -1., derivative);
    norm = dg::blas2::dot( derivative, vol3d, derivative);
    name  = "directLapST"; // works as well as directLap
    std::cout <<"    "<<name<<":" <<std::setw(18-name.size())
              <<" "<<sqrt(norm/sol)<<"\n"
              <<"    "<<name+"_vol:"<<std::setw(30-name.size())
              <<" "<<vol<<"\n";
    ///##########################################################///
    std::cout << "# TEST VOLUME FORMS\n";
    double volume = dg::blas1::dot( 1., dsFAST.sqrtG());
    double volumeM = dg::blas1::dot( 1., dsFAST.sqrtGm());
    double volumeP = dg::blas1::dot( 1., dsFAST.sqrtGp());
    std::cout << "volume_error:\n";
    std::cout <<"    minus:"<<std::setw(13)<<" "<<fabs(volumeM-volume)/volume<<"\n";
    std::cout <<"    plus:" <<std::setw(14)<<" "<<fabs(volumeP-volume)/volume<<"\n";


    dg::DVec f(g3d.size(), 1.), temp1(f), temp2(f), temp3(f);
    dsFAST(dg::geo::einsPlus, f, temp1);

    dg::blas1::pointwiseDot( dsFAST.sqrtG(), temp1, temp3);
    dsFAST(dg::geo::einsPlusT, temp3, temp2);
    dg::blas1::pointwiseDivide( temp2, dsFAST.sqrtGm(), temp2);
    dg::blas1::axpby( 1., temp2, -1., 1., temp2);
    dsFAST(dg::geo::einsPlus, temp2, temp3);

    double error = dg::blas2::dot( temp3, vol, temp3);
    //norm = dg::blas2::dot( 1., vol, 1.);
    norm = dg::blas2::dot( temp1, vol, temp1);
    std::cout <<"    Inv:"<<std::setw(15)<<" "<<sqrt(error/norm)<<"\n";

    return 0;
}
