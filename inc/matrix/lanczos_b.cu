#include "hip/hip_runtime.h"
#define DG_DEBUG
#include <iostream>
#include <iomanip>
#include "backend/timer.h"
#include "lanczos.h"
#include "helmholtz.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
const double alpha = -1;
const double m = 1.;
const double n = 1.;

double lhs( double x, double y) {return sin(m*x)*sin(n*y);}
double rhs( double x, double y){ return (1.-(m*m+n*n)*alpha)*sin(m*x)*sin(n*y);}

using Matrix = dg::DMatrix;
using Container = dg::DVec;
using HDiaMatrix = cusp::dia_matrix<int, double, cusp::host_memory>;
using HCooMatrix = cusp::coo_matrix<int, double, cusp::host_memory>;

int main(int argc, char * argv[])
{
    dg::Timer t;
    unsigned n, Nx, Ny;
    std::cout << "# Type n, Nx and Ny\n";
    std::cin >> n >> Nx >> Ny;
    std::cout <<"# You typed\n"
              <<"n:  "<<n<<"\n"
              <<"Nx: "<<Nx<<"\n"
              <<"Ny: "<<Ny<<std::endl;
    unsigned max_iter;
    std::cout << "# Type in max_iter and eps\n"; 
    double eps = 1e-6; 
    std::cin >> max_iter>> eps;
    std::cout <<"# You typed\n"
              <<"max_iter:  "<<max_iter<<"\n"
              <<"eps: "<<eps <<std::endl;  
    dg::CartesianGrid2d grid( 0., lx, 0, ly, n, Nx, Ny, bcx, bcy);
    
    const Container w2d = dg::create::weights( grid);
    const Container v2d = dg::create::inv_weights( grid);
        
    Container x = dg::evaluate( lhs, grid), b(x), zero(x), one(x), error(x),  helper(x), xexac(x);
    Container bexac = dg::evaluate( rhs, grid);
    dg::blas1::scal(zero, 0.0);
    one = dg::evaluate(dg::one, grid);
    dg::Helmholtz<dg::CartesianGrid2d, Matrix, Container> A( grid, alpha, dg::centered); //not_normed
    
    {
        t.tic();
        dg::Lanczos< Container > lanczos(x, max_iter);
        t.toc();
        std::cout << "# Lanczos creation took "<< t.diff()<<"s   \n";

        HDiaMatrix T; 
        std::cout << "Lanczos:\n";
       
        t.tic();
        T = lanczos( A, x, b, eps, true); 
        dg::blas2::symv( v2d, b, b);     //normalize
        t.toc();
        
        std::cout << "    iter: "<< lanczos.get_iter() << "\n";
        std::cout << "    time: "<< t.diff()<<"s \n";
        dg::blas1::axpby(-1.0, bexac, 1.0, b,error);
        std::cout << "    # Relative error between b=||x||_2 V^T T e_1 and b: \n";   
        std::cout << "    error: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bexac)) << " \n";   

        std::cout << "\nM-Lanczos:\n";
        x = dg::evaluate( lhs, grid);
        t.tic();
        T = lanczos(A, x, b, v2d, w2d, eps, true); 
        t.toc();
        std::cout << "    iter: "<< lanczos.get_iter() << "\n";
        std::cout << "    time: "<< t.diff()<<"s \n";
        dg::blas1::axpby(-1.0, bexac, 1.0, b,error);
        std::cout << "    # Relative error between b=||x||_M V^T T e_1 and b: \n";  
        std::cout << "    error: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bexac)) << " \n";   

    } 
    {
        std::cout << "\nM-CG: \n";
        t.tic();
        dg::MCG<Container> mcg(x, max_iter);
        t.toc();
        std::cout << "#    M-CG creation took "<< t.diff()<<"s   \n";
//         dg::blas1::scal(x, 0.0); //initialize with zero
//         dg::blas1::scal(x, 0.0); //initialize with zero
        x =    dg::evaluate(dg::one, grid);
        dg::blas1::scal(x,1000.0); //initialize not with zero

        dg::blas2::symv(w2d, bexac, b); //multiply weights
        t.tic();
        HDiaMatrix T = mcg(A, x, b, v2d, w2d, eps, 1., true); 
        t.toc();
        


        dg::blas1::axpby(-1.0, xexac, 1.0, x, error);
        std::cout << "    iter: "<< mcg.get_iter() << "\n";
        std::cout << "    time: "<< t.diff()<<"s \n";
        std::cout << "    # Relative error between x= R T^{-1} e_1 and x: \n";
        std::cout << "    error: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, xexac)) << " \n";
    }

    
    return 0;
}
