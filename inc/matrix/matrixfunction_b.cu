#include "hip/hip_runtime.h"
// #define DG_DEBUG

#include <iostream>
#include <iomanip>

#include "dg/algorithm.h"
#include "lanczos.h"
#include "mcg.h"
#include "matrixfunction.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
const double m=1.;
const double n=1.;
const double alpha = -1.;
const double ell_fac = -alpha*(m*m+n*n);
const double helm_fac = 1.+ ell_fac;

double lhs( double x, double y){ return sin(x*m)*sin(y*n);}

using Matrix = dg::DMatrix;
using Container = dg::DVec;

int main(int argc, char * argv[])
{
    dg::Timer t;

    unsigned n, Nx, Ny;
    std::cout << "# Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout <<"# You typed\n"
              <<"n:  "<<n<<"\n"
              <<"Nx: "<<Nx<<"\n"
              <<"Ny: "<<Ny<<std::endl;
    unsigned iter = 0;

    unsigned max_iter = 1;
    std::cout << "# Type max_iter of tridiagonalization (500) ?\n";
    std::cin >> max_iter ;
    std::cout << "# Type in eps of tridiagonalization (1e-7)\n";
    double eps = 1e-7; //# of pcg iter increases very much if
    std::cin >> eps;
    std::cout <<"# You typed\n"
              <<"max_iter: "<<max_iter<<"\n"
              <<"eps: "<<eps<<std::endl;

    double erel = 0;

    dg::Grid2d g( 0, lx, 0, ly,n, Nx, Ny, bcx, bcy);
    const Container w2d = dg::create::weights( g);
    double max_weights = dg::blas1::reduce(w2d, 0., dg::AbsMax<double>() );
    double min_weights = dg::blas1::reduce(w2d, max_weights, dg::AbsMin<double>() );
    std::cout << "#   min(W)  = "<<min_weights <<"  max(W) = "<<max_weights << "\n";
    const double kappa = sqrt(max_weights/min_weights); //condition number
    dg::Helmholtz<dg::CartesianGrid2d, Matrix, Container> A( g, alpha, dg::centered);
    dg::mat::UniversalLanczos<Container> lanczos( A.weights(), 20);
    auto T = lanczos.tridiag( A, A.weights(), A.weights());
    auto extremeEVs = dg::mat::compute_extreme_EV( T);
    double EVmin = extremeEVs[0];
    double EVmax = extremeEVs[1];

    std::vector< std::function<double (double)>> funcs{
        [](double x) { return sqrt(x);},
        [](double x) { return 1./sqrt(x);},
        [](double x) { return 1./exp(x);},
        [](double x) {
            return 1./boost::math::cyl_bessel_i(0, x);},
        [](double x){
            return 1./exp(x)/boost::math::cyl_bessel_i(0, x);
        },
        [](double x) { return 1./x;}
    };
    std::vector<std::string> outs = {"Sqrt", "Inv-Sqrt", "Inv-Exp",
        "Inv-Bessel", "Inv-Gamma", "Inverse"};
    for( unsigned u=0; u<funcs.size(); u++)
    {
        std::cout << "\n#Compute x = "<<outs[u]<<"(1+ alpha Delta) b " << std::endl;

        Container x = dg::evaluate(lhs, g), x_exac(x), b(x), error(x);
        dg::blas1::scal(x_exac, funcs[u](helm_fac));

        double res_fac = kappa*funcs[u](EVmin);
        std::cout << "#   min(EV) = "<<EVmin <<"  max(EV) = "<<EVmax << "\n";
        std::cout << "#   kappa   = "<<kappa <<"\n";
        std::cout << "#   res_fac = "<<res_fac<< "\n";
        std::cout << outs[u] << "\n";
        dg::mat::UniversalLanczos<Container> krylovfunceigen( x, max_iter);
        t.tic();
        auto func = dg::mat::make_FuncEigen_Te1( funcs[u]);
        iter = krylovfunceigen.solve(x, func, A, b, w2d, eps, 1.,
                "residual", res_fac);
        t.toc();
        double time = t.diff();

        dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
        erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));

        std::cout << "    residual-time: "<<time<<"s \n";
        std::cout << "    residual-error: "<<erel  << "\n";
        std::cout << "    residual-iter: "<<std::setw(3)<<iter << "\n";

        dg::mat::MCGFuncEigen<Container> mcgfunceigen( x, max_iter);
        t.tic();
        iter = mcgfunceigen(x, funcs[u], A, b, w2d, eps, 1.,
                res_fac);
        t.toc();
        time = t.diff();

        dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
        erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));

        std::cout << "    mcg-time: "<<time<<"s \n";
        std::cout << "    mcg-error: "<<erel  << "\n";
        std::cout << "    mcg-iter: "<<std::setw(3)<<iter << "\n";

        t.tic();
        iter = krylovfunceigen.solve(x, func, A, b, w2d, eps, 1.,
                "universal");
        t.toc();
        time = t.diff();

        dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
        erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));

        std::cout << "    universal-time: "<<time<<"s \n";
        std::cout << "    universal-error: "<<erel  << "\n";
        std::cout << "    universal-iter: "<<std::setw(3)<<iter << "\n";
    }
    return 0;
}
