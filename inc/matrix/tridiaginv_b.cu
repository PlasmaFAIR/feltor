#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "tridiaginv.h"

#include "backend/timer.h"
#include <cusp/transpose.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/elementwise.h>
#include <cusp/blas/blas.h>
#include "cg.h"
#include "lgmres.h"
#include "bicgstabl.h"
#include <cusp/print.h>

using value_type = double;
using memory_type = cusp::host_memory;
using CooMatrix =  cusp::coo_matrix<int, double, memory_type>;
using DiaMatrix =  cusp::dia_matrix<int, double, memory_type>;
using Container = dg::HVec;

double mu(double s, unsigned i, unsigned n) { 
    return (1.0+1.0/s*(1.0-1.0/pow(1.0 + s,n-i-1.0)));
}

int main()
{
    dg::Timer t;
    unsigned size = 50;
    std::cout << "#Specify size of vectors (50)\n";
    std::cin >> size;
    unsigned max_outer =300;
    unsigned max_inner = 300;
    unsigned restarts = 30000;
//     std::cout << "# max_outer, max_inner and restarts of lgmres (30,10,10000) \n";
//     std::cin >> max_outer >> max_inner >> restarts;
    
    std::cout << "#Constructing and filling vectors\n";
    std::vector<value_type> a(size,1.);
    std::vector<value_type> b(size,1.);
    std::vector<value_type> c(size,1.);
    std::vector<value_type> a_sym(size,1.);
    std::vector<value_type> b_sym(size,1.);
    double s= 1.1;
    for (unsigned i=0;i<size; i++)
    {
        //vectors of non-symmetric tridiagonal matrix
        a[i] = 1.0;
        b[i] = -1.0/(2.0+s);
        c[i] = -(1.0+s)/(2.0+s);
        //vectors of symmetric tridiagonal matrix
        if (i<size-1) {
            a_sym[i] = 4.0*(i+1)*(i+1)*(i+1)/(4.0*(i+1)*(i+1)-1.0);
        }
        else {
            a_sym[i] = size*size/(2.0*size-1.0);
        }        
        b_sym[i] = -1.0*(i+1)*((i+1)+1.0)/(2.0*(1+i)+1.0);
    }
    std::cout << "#Constructing and filling containers\n";
    const Container d(size,1.);
    Container x(size,0.), x_symsol(x), x_sol(x), err(x);
    std::cout << "#Constructing Matrix inversion and linear solvers\n";
    value_type eps= 1e-20;
    t.tic();
    dg::CG <Container> pcg( x,  size*size+1);
    t.toc();
    std::cout << "#Construction of CG took "<< t.diff()<<"s \n";
    t.tic();    
    dg::LGMRES <Container> lgmres( x, max_outer, max_inner, restarts);
    t.toc();
    std::cout << "#Construction of LGMRES took "<< t.diff()<<"s \n";
    t.tic();    
    dg::BICGSTABl <Container> bicg( x,size*size,4);
    t.toc();
    std::cout << "#Construction of BICGSTABl took "<< t.diff()<<"s \n";
    t.tic();
    dg::TridiagInvDF<Container, DiaMatrix, CooMatrix> tridiaginvDF(a);
    t.toc();
    std::cout << "#Construction of Tridiagonal inversion DF routine took "<< t.diff()<<"s \n";
    t.tic();
    dg::TridiagInvD<Container, DiaMatrix, CooMatrix> tridiaginvD(a);
    t.toc();
    std::cout << "#Construction of Tridiagonal inversion D routine took "<< t.diff()<<"s \n";
    
    //Create Tridiagonal and fill matrix
    DiaMatrix T, Tsym; 
    T.resize(size, size, 3*size-2, 3);
    T.diagonal_offsets[0] = -1;
    T.diagonal_offsets[1] =  0;
    T.diagonal_offsets[2] =  1;
    Tsym.resize(size, size, 3*size-2, 3);
    Tsym.diagonal_offsets[0] = -1;
    Tsym.diagonal_offsets[1] =  0;
    Tsym.diagonal_offsets[2] =  1;
    
    for( unsigned i=0; i<size-1; i++)
    {
        T.values(i,1)   =  a[i];  // 0 diagonal
        T.values(i+1,0) =  c[i];  // -1 diagonal
        T.values(i,2)   =  b[i];  // +1 diagonal //dia_rows entry works since its outside of matrix
        Tsym.values(i,1)   =  a_sym[i];  // 0 diagonal
        Tsym.values(i+1,0) =  b_sym[i];  // -1 diagonal
        Tsym.values(i,2)   =  b_sym[i];  // +1 diagonal //dia_rows entry works since its outside of matrix
    }
    T.values(size-1,1) =  a[size-1];
    Tsym.values(size-1,1) =  a_sym[size-1];
    
    //Create Inverse of tridiagonal matrix
    CooMatrix Tinv, Tsyminv, Tinv_sol, Tsyminv_sol;
    Tinv_sol.resize(size, size,  size* size);
    Tsyminv_sol.resize(size, size,  size* size);    
    for( unsigned i=0; i<size; i++) //row index
    {   
        for( unsigned j=0; j<size; j++) //column index
        {   
            Tinv_sol.row_indices[i*size+j]    = i;
            Tinv_sol.column_indices[i*size+j] = j; 
            Tsyminv_sol.row_indices[i*size+j]    = i;
            Tsyminv_sol.column_indices[i*size+j] = j; 
            if (i>= j) 
            {
                Tinv_sol.values[i*size+j] = (2.0+s)/(1.0+s)*mu(s,i+1,size+1)*mu(s,size+1-(j+1),size+1)/mu(s,0,size+1);
                Tsyminv_sol.values[i*size+j] = (j+1.0)/(i+1.0);
            }
            else
            {
                Tsyminv_sol.values[i*size+j] = (i+1.0)/(j+1.0);
            }
            
        }
    }
    for( unsigned i=0; i<size; i++) //row index
    {   
        for( unsigned j=0; j<size; j++) //column index
        {   
            if (i<j) 
            {
                Tinv_sol.values[i*size+j] = pow(1.0/(1.0+s),j-i)*Tinv_sol.values[j*size+i];
            }
            
        }
    }
    dg::blas2::gemv(Tinv_sol, d, x_sol);
    dg::blas2::gemv(Tsyminv_sol, d, x_symsol);

    //Do inversions
    std::cout << "####Compute inverse of symmetric tridiagonal matrix\n";
    std::cout << "CG:" << std::endl;
    dg::blas1::scal(x, 0.);
    t.tic();
    unsigned number = pcg( Tsym, x, d, d, eps);
    if(  number == pcg.get_max())
        throw dg::Fail( eps);
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_symsol, err );
    std::cout << "    time: "<< t.diff()<<"s \n";
    std::cout << "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_symsol,x_symsol)) << "\n";
    std::cout << "InvtridiagDF(v_sym):" << std::endl;
    t.tic();
    Tsyminv = tridiaginvDF(a_sym,b_sym,b_sym);
    t.toc();
    dg::blas2::gemv(Tsyminv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_symsol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_symsol,x_symsol)) << "\n";
    std::cout << "InvtridiagDF(Tsym):" << std::endl;
    t.tic();
    Tsyminv = tridiaginvDF(Tsym);
    t.toc();
    dg::blas2::gemv(Tsyminv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_symsol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_symsol,x_symsol)) << "\n";
    std::cout <<  "    #error_rel in T_{m,1}: " << abs(Tsyminv.values[size-1] - Tsyminv_sol.values[size-1])/abs(Tsyminv_sol.values[size-1]) << "\n";
    std::cout << "InvtridiagD(v_sym):" << std::endl;
    t.tic();
    Tsyminv = tridiaginvD(a_sym,b_sym,b_sym);
    t.toc();
    dg::blas2::gemv(Tsyminv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_symsol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_symsol,x_symsol)) << "\n";
    std::cout << "InvtridiagD(Tsym):" << std::endl;
    t.tic();
    Tsyminv = tridiaginvD(Tsym);
    t.toc();
    dg::blas2::gemv(Tsyminv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_symsol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_symsol,x_symsol)) << "\n";
    std::cout <<  "    #error_rel in T_{m,1}: " << abs(Tsyminv.values[size-1] - Tsyminv_sol.values[size-1])/abs(Tsyminv_sol.values[size-1]) << "\n";
    

    std::cout << "\n####Compute inverse of non-symmetric tridiagonal matrix\n";
    std::cout << "lGMRES:" << std::endl;
    dg::blas1::scal(x, 0.);
    t.tic();
    number = lgmres.solve( T, x, d , d, d, eps, 1);    
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";  
    std::cout << "BICGSTABl:" << std::endl;

    dg::blas1::scal(x, 0.);
    t.tic();
    number = bicg.solve( T, x, d , d, d, eps, 1);    
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";     
    std::cout << "InvtridiagDF(v):" << std::endl;
    t.tic();
    Tinv = tridiaginvDF(a,b,c);
    t.toc();
    dg::blas2::gemv(Tinv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";
    std::cout << "InvtridiagDF(T):" << std::endl;
    t.tic();
    Tinv = tridiaginvDF(T);
    t.toc();
    dg::blas2::gemv(Tinv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";    
    std::cout << "InvtridiagD(v):" << std::endl;
    t.tic();
    Tinv = tridiaginvD(a,b,c);
    t.toc();
    dg::blas2::gemv(Tinv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";
    std::cout << "InvtridiagD(T):" << std::endl;
    t.tic();
    Tinv = tridiaginvD(T);
    t.toc();
    dg::blas2::gemv(Tinv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "    time: "<< t.diff()<<"s \n";
    std::cout <<  "    error_rel: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";     

    return 0;
}
