#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "json/json.h"

#include "dg/algorithm.h"
#include "dg/geometries/geometries.h"
#include "dg/file/json_utilities.h"

#include "parameters.h"
#define DG_MANUFACTURED
//Change here to selectively test parallel and perp parts
#define FELTORPARALLEL 1
#define FELTORPERP 0

#include "manufactured.h"
#include "feltor.h"
#include "implicit.h"

int main( int argc, char* argv[])
{
    dg::file::WrappedJsonValue js( dg::file::error::is_throw);
    std::string inputfile = argc == 1 ? "input.json" : argv[1];
    dg::file::file2Json( inputfile, js.asJson(), dg::file::comments::are_forbidden);

    const feltor::Parameters p( js);
    std::cout << js.asJson() <<std::endl;
    const double R_0 = 10;
    const double I_0 = 20; //q factor at r=1 is I_0/R_0
    const double a  = 1; //small radius
    dg::CylindricalGrid3d grid( R_0-a, R_0+a, -a, a, 0, 2.*M_PI,
        p.n, p.Nx, p.Ny, p.Nz, p.bcxN, p.bcyN, dg::PER);
    dg::DVec w3d = dg::create::volume( grid);

    //create RHS
    std::cout << "Initialize explicit" << std::endl;
    dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
    feltor::Explicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec> feltor( grid, p, mag);

    feltor::manufactured::Ne ne{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                 p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};
    feltor::manufactured::Ni ni{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                 p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};
    feltor::manufactured::Ue ue{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};
    feltor::manufactured::Ui ui{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                 p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};
    feltor::manufactured::Phie phie{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                     p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};
    feltor::manufactured::Phii phii{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                     p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};
    feltor::manufactured::A aa{ p.mu[0],p.mu[1],p.tau[0],p.tau[1],p.eta,
                                p.beta,p.nu_perp,p.nu_parallel[0],p.nu_parallel[1]};

    dg::DVec R = dg::pullback( dg::cooX3d, grid);
    dg::DVec Z = dg::pullback( dg::cooY3d, grid);
    dg::DVec P = dg::pullback( dg::cooZ3d, grid);
    std::array<dg::DVec,2> phi{R,R}, sol_phi{phi};
    std::array<std::array<dg::DVec,2>,2> y0{phi,phi}, sol{y0};
    dg::DVec apar{R}, sol_apar{apar};
    dg::blas1::evaluate( y0[0][0], dg::equals(), ne, R,Z,P,0);
    dg::blas1::evaluate( y0[0][1], dg::equals(), ni, R,Z,P,0);
    dg::blas1::evaluate( y0[1][0], dg::equals(), ue, R,Z,P,0);
    dg::blas1::evaluate( y0[1][1], dg::equals(), ui, R,Z,P,0);
    dg::blas1::evaluate( apar, dg::equals(), aa, R,Z,P,0);
    dg::blas1::plus(y0[0][0],-1); //ne-1
    dg::blas1::plus(y0[0][1],-1); //Ni-1
    dg::blas1::axpby(1./p.mu[0], apar, 1., y0[1][0]); //we=ue+1/mA
    dg::blas1::axpby(1./p.mu[1], apar, 1., y0[1][1]); //Wi=Ui+1/mA

    dg::ExplicitMultistep< std::array<std::array<dg::DVec,2>,2 > > mp(p.tableau, y0);
    double time = 0, TMAX = 0.1;
    mp.init( feltor, time, y0, p.dt);
    while( time < TMAX)
    {
        try{
            mp.step( feltor, time, y0);
        }
        catch( dg::Fail& fail) {
            std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
            std::cerr << "Does Simulation respect CFL condition?\n";
            return -1;
        }
        std::cout << "#Time "<<time<<std::endl;
    }
    //now compare stuff
    dg::blas1::evaluate( sol[0][0], dg::equals(), ne, R,Z,P,time);
    dg::blas1::evaluate( sol[0][1], dg::equals(), ni, R,Z,P,time);
    dg::blas1::evaluate( sol[1][0], dg::equals(), ue, R,Z,P,time);
    dg::blas1::evaluate( sol[1][1], dg::equals(), ui, R,Z,P,time);
    dg::blas1::evaluate( sol_apar, dg::equals(), aa, R,Z,P,time);
    dg::blas1::evaluate( sol_phi[0], dg::equals(),phie,R,Z,P,time);
    dg::blas1::evaluate( sol_phi[1], dg::equals(),phii,R,Z,P,time);
    dg::blas1::plus(sol[0][0],-1); //ne-1
    dg::blas1::plus(sol[0][1],-1); //Ni-1
    const std::array<std::array<dg::DVec,2>,2>& num = feltor.fields();
    const std::array<dg::DVec,2>& num_phi = feltor.potentials();
    const dg::DVec& num_apar = feltor.aparallel();
    double normne = sqrt(dg::blas2::dot( w3d, sol[0][0]));
    double normni = sqrt(dg::blas2::dot( w3d, sol[0][1]));
    double normue = sqrt(dg::blas2::dot( w3d, sol[1][0]));
    double normui = sqrt(dg::blas2::dot( w3d, sol[1][1]));
    double normphie = sqrt(dg::blas2::dot( w3d, sol_phi[0]));
    double normphii = sqrt(dg::blas2::dot( w3d, sol_phi[1]));
    double normapar = sqrt(dg::blas2::dot( w3d, sol_apar));
    dg::blas1::axpby( 1., y0[0][0], -1.,sol[0][0]);
    dg::blas1::axpby( 1., y0[0][1], -1.,sol[0][1]);
    dg::blas1::axpby( 1., num[1][0], -1.,sol[1][0]);
    dg::blas1::axpby( 1., num[1][1], -1.,sol[1][1]);
    dg::blas1::axpby( 1., num_phi[0], -1.,sol_phi[0]);
    dg::blas1::axpby( 1., num_phi[1], -1.,sol_phi[1]);
    dg::blas1::axpby( 1., num_apar, -1.,sol_apar);
    std::cout<<std::scientific;
    std::cout <<"           rel. Error\tNorm: \n"
              //<<"    Time: "<<time<<"\n"
              <<"    ne:   "<<sqrt(dg::blas2::dot( w3d, sol[0][0]))/normne<<"\t"<<normne<<"\n"
              <<"    ni:   "<<sqrt(dg::blas2::dot( w3d, sol[0][1]))/normni<<"\t"<<normni<<"\n"
              <<"    ue:   "<<sqrt(dg::blas2::dot( w3d, sol[1][0]))/normue<<"\t"<<normue<<"\n"
              <<"    ui:   "<<sqrt(dg::blas2::dot( w3d, sol[1][1]))/normui<<"\t"<<normui<<"\n"
              <<"    phie: "<<sqrt(dg::blas2::dot( w3d,sol_phi[0]))/normphie<<"\t"<<normphie<<"\n"
              <<"    phii: "<<sqrt(dg::blas2::dot( w3d,sol_phi[1]))/normphii<<"\t"<<normphii<<"\n"
              <<"    apar: "<<sqrt(dg::blas2::dot( w3d,sol_apar))/normapar<<"\t"<<normapar<<"\n";

    return 0;

}
