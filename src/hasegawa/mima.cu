#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"

#include "mima.cuh"
#include "../toefl/parameters.h"
#include "dg/file/json_utilities.h"

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    std::stringstream title;
    Json::Value js;
    if( argc == 1)
        dg::file::file2Json( "input.json", js, dg::file::comments::are_discarded);
    else if( argc == 2)
        dg::file::file2Json( argv[1], js, dg::file::comments::are_discarded);
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    const Parameters p( js);
    p.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    dg::file::file2Json( "window_params.json", js, dg::file::comments::are_discarded);
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["width"].asDouble(), js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), js["cols"].asDouble());
    /////////////////////////////////////////////////////////////////////////
    dg::CartesianGrid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS 
    bool mhw = ( p.equations == "fullF");
    mima::Mima< dg::CartesianGrid2d, dg::DMatrix, dg::DVec > mima( grid, p.kappa, p.tau, p.eps_pol, mhw); 
    dg::DVec one( grid.size(), 1.);
    //create initial vector
    dg::Gaussian gaussian( p.posX*grid.lx(), p.posY*grid.ly(), p.sigma, p.sigma, p.amp); //gaussian width is in absolute values
    dg::Vortex vortex( p.posX*grid.lx(), p.posY*grid.ly(), 0, p.sigma, p.amp);

//     dg::DVec phi = dg::evaluate( vortex, grid), omega( phi), y0(phi), y1(phi);
    dg::DVec phi = dg::evaluate( gaussian, grid), omega( phi), y0(phi), y1(phi);
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> laplaceM( grid,  dg::centered);
    dg::blas2::gemv( laplaceM, phi, omega);
    dg::blas1::axpby( 1., phi, 1., omega, y0);

    dg::DVec w2d( dg::create::weights( grid));
    if( p.bc_x == dg::PER && p.bc_y == dg::PER)
    {
        double meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
        std::cout << "Mean Mass is "<<meanMass<<"\n";
        dg::blas1::axpby( -meanMass, one, 1., y0);
    }
    dg::Karniadakis<dg::DVec > ab( y0, y0.size(), p.eps_time);
    mima::Diffusion<dg::CartesianGrid2d,dg::DMatrix,dg::DVec> diffusion( grid, p.nu);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    ab.init( mima, diffusion, time, y0, p.dt);
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        if( p.bc_x == dg::PER && p.bc_y == dg::PER)
        {
            double meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
            std::cout << "Mean Mass is "<<meanMass<<"\n";
        }
        //transform field to an equidistant grid
        dvisual = mima.potential();

        dg::assign( dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        dg::blas2::gemv( laplaceM, mima.potential(), y1);
        dg::assign( y1, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"omega / "<<colors.scale()<<"\t";
        title << std::fixed;
        title << " &&   time = "<<time;
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            if( p.bc_x == dg::PER && p.bc_y == dg::PER)
            {
                double meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
                dg::blas1::axpby( -meanMass, one, 1., y0);
                meanMass = dg::blas2::dot( y0, w2d, one)/(double)(p.lx*p.ly);
                dg::blas1::axpby( -meanMass, one, 1., y0);
            }

            try{ ab.step( mima, diffusion, time, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
